#include "hip/hip_runtime.h"
/**
 * @file MxBoundaryConditions_cuda.cu
 * @author T.J. Sego, Ph.D.
 * @brief Defines boundary condition kernels on CUDA-supporting GPUs
 * @date 2021-11-24
 * 
 */

// TODO: improve error handling in MxBoundaryConditions_cuda

#include "MxBoundaryConditions_cuda.h"

#include "engine.h"


// MxBoundaryConditionCUDA


__host__ 
MxBoundaryConditionCUDA::MxBoundaryConditionCUDA(const MxBoundaryCondition &_bc) {
    this->normal = make_float3(_bc.normal[0], _bc.normal[1], _bc.normal[2]);
    this->velocity = make_float3(_bc.velocity[0], _bc.velocity[1], _bc.velocity[2]);
    this->radius = _bc.radius;

    MxPotential *p;
    
    size_t size_pots = sizeof(MxPotentialCUDA) * engine_maxnrtypes;
    if(hipMalloc(&this->pots, size_pots) != hipSuccess) {
        printf("Boundary condition allocation failed: %s\n", hipGetErrorString(hipPeekAtLastError()));
        return;
    }

    this->pots_h = (MxPotentialCUDA*)malloc(size_pots);
    for(int typeId = 0; typeId < engine_maxnrtypes; typeId++) { 
        p = _bc.potenntials[typeId];
        if(p != NULL) 
            this->pots_h[typeId] = MxPotentialCUDA(*p);
        else 
            this->pots_h[typeId] = MxPotentialCUDA();
    }

    if(hipMemcpy(this->pots, this->pots_h, size_pots, hipMemcpyHostToDevice) != hipSuccess)
        printf("Boundary condition copy H2D failed: %s\n", hipGetErrorString(hipPeekAtLastError()));
}

__host__ 
void MxBoundaryConditionCUDA::finalize() {
    for(int typeId = 0; typeId < engine_maxnrtypes; typeId++) {
        auto p = this->pots_h[typeId];
        p.finalize();
    }

    free(this->pots_h);
    
    if(hipFree(this->pots) != hipSuccess) 
        printf("Boundary condition finalize failed: %s\n", hipGetErrorString(hipPeekAtLastError()));
}


// MxBoundaryConditionsCUDA


__host__ 
MxBoundaryConditionsCUDA::MxBoundaryConditionsCUDA(const MxBoundaryConditions &_bcs) {
    size_t size_bcs = sizeof(MxBoundaryConditionCUDA) * 6;

    if(hipMalloc(&this->bcs, size_bcs) != hipSuccess) {
        printf("Boundary conditions allocation failed: %s\n", hipGetErrorString(hipPeekAtLastError()));
        return;
    }

    this->bcs_h = (MxBoundaryConditionCUDA*)malloc(size_bcs);

    this->bcs_h[0] = MxBoundaryConditionCUDA(_bcs.left);
    this->bcs_h[1] = MxBoundaryConditionCUDA(_bcs.right);
    this->bcs_h[2] = MxBoundaryConditionCUDA(_bcs.front);
    this->bcs_h[3] = MxBoundaryConditionCUDA(_bcs.back);
    this->bcs_h[4] = MxBoundaryConditionCUDA(_bcs.bottom);
    this->bcs_h[5] = MxBoundaryConditionCUDA(_bcs.top);

    if(hipMemcpy(this->bcs, this->bcs_h, size_bcs, hipMemcpyHostToDevice) != hipSuccess)
        printf("Boundary conditions copy H2D failed: %s\n", hipGetErrorString(hipPeekAtLastError()));
}

__host__ 
void MxBoundaryConditionsCUDA::finalize() {
    for(int bcId = 0; bcId < 6; bcId++)
        this->bcs_h[bcId].finalize();

    free(this->bcs_h);

    if(hipFree(this->bcs) != hipSuccess) 
        printf("Boundary conditions finalize failed: %s\n", hipGetErrorString(hipPeekAtLastError()));
}
