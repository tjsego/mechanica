/**
 * @file MxPotential_cuda.cu
 * @author T.J. Sego, Ph.D.
 * @brief Defines potential kernels on CUDA-supporting GPUs
 * @date 2021-11-24
 * 
 */

#include "MxPotential_cuda.h"


MxPotential MxToCUDADevice(const MxPotential &p) {
    MxPotential p_d(p);

    // Alloc and copy coefficients
    if(hipMalloc(&p_d.c, sizeof(FPTYPE) * (p.n + 1) * potential_chunk) != hipSuccess) {
        mx_error(E_FAIL, hipGetErrorString(hipPeekAtLastError()));
        return p_d;
    }
    if(hipMemcpy(p_d.c, p.c, sizeof(FPTYPE) * (p.n + 1) * potential_chunk, hipMemcpyHostToDevice) != hipSuccess) {
        mx_error(E_FAIL, hipGetErrorString(hipPeekAtLastError()));
        return p_d;
    }

    if(p.pca != NULL) { 
        MxPotential pca_d = MxToCUDADevice(*p.pca);
        if(hipMalloc(&p_d.pca, sizeof(MxPotential)) != hipSuccess) 
            mx_error(E_FAIL, "pca malloc failed!");
        if(hipMemcpy(p_d.pca, &pca_d, sizeof(MxPotential), hipMemcpyHostToDevice) != hipSuccess) 
            mx_error(E_FAIL, "pca load H2D failed!");
    }
    else 
        p_d.pca = NULL;
    if(p.pcb != NULL) { 
        MxPotential pcb_d = MxToCUDADevice(*p.pcb);
        if(hipMalloc(&p_d.pcb, sizeof(MxPotential)) != hipSuccess) 
            mx_error(E_FAIL, "pcb malloc failed!");
        if(hipMemcpy(p_d.pcb, &pcb_d, sizeof(MxPotential), hipMemcpyHostToDevice) != hipSuccess) 
            mx_error(E_FAIL, "pcb load H2D failed!");
    } 
    else 
        p_d.pcb = NULL;

    return p_d;
}

__host__ __device__ 
void Mx_cudaFree(MxPotential *p) {
    if(p == NULL || p->flags & POTENTIAL_NONE) 
        return;
    
    if(p->pca != NULL) {
        MxPotential *pca;
        if(hipMemcpy(pca, p->pca, sizeof(MxPotential), hipMemcpyDeviceToHost) != hipSuccess) 
            printf("%s\n", "pca load D2H failed!");
        Mx_cudaFree(pca);
    }
    if(p->pcb != NULL) {
        MxPotential *pcb;
        if(hipMemcpy(pcb, p->pcb, sizeof(MxPotential), hipMemcpyDeviceToHost) != hipSuccess)
            printf("%s\n", "pcb load D2H failed!");
        Mx_cudaFree(pcb);
    }

    hipFree(p->c);
    p->c = NULL;
}
