#include "hip/hip_runtime.h"
/*******************************************************************************
 * This file is part of mdcore.
 * Coypright (c) 2012 Pedro Gonnet (pedro.gonnet@durham.ac.uk)
 * 
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU Lesser General Public License as published
 * by the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 * 
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 * 
 * You should have received a copy of the GNU Lesser General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 * 
 ******************************************************************************/

// TODO: implement hook for potentials by particles
// TODO: implement support for clusters
// TODO: implement boundary conditions

/* Include configuratin header */
#include <mdcore_config.h>

/* Include some standard header files */
#include <stdlib.h>
#include <stdio.h>
#include <pthread.h>
#include <math.h>
#include <float.h>
#include <string.h>
#include <limits.h>

/* Include headers for overloaded vector functions. */
#include "cutil_math.h"

#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

/* Include some conditional headers. */
#ifdef HAVE_MPI
    #include <mpi.h>
#endif

/* Force single precision. */
#ifndef FPTYPE_SINGLE
    #define FPTYPE_SINGLE 1
#endif

/* Disable vectorization for the nvcc compiler's sake. */
#undef __SSE__
#undef __SSE2__
#undef __ALTIVEC__
#undef __AVX__

/* Include local headers */
#include "cycle.h"
#include "errs.h"
#include "fptype.h"
#include "lock.h"
#include "MxParticle.h"
#include "space_cell.h"
#include "space.h"
#include "task.h"
#include "MxPotential.h"
#include "DissapativeParticleDynamics.hpp"
#include "engine.h"
#include "runner_cuda.h"

#ifndef CPU_TPS
#include <ctime>
#define CPU_TPS CLOCKS_PER_SEC
#endif

/* the error macro. */
#define error(id)				( engine_err = errs_register( id , engine_err_msg[-(id)] , __LINE__ , __FUNCTION__ , __FILE__ ) )
#define cuda_error(id)			( engine_err = errs_register( id , hipGetErrorString(hipGetLastError()) , __LINE__ , __FUNCTION__ , __FILE__ ) )

struct MxParticleCUDA;

/* The constant null potential. */
__constant__ struct MxPotential *potential_null_cuda = NULL;

/* The number of cells and pairs. */
__constant__ int cuda_nr_cells = 0;

/* The parts (non-texture access). */
__constant__ MxParticleCUDA *cuda_parts;
__constant__ int cuda_nr_parts;
__constant__ int cuda_size_parts;

/* Diagonal entries and potential index lookup table. */
__constant__ unsigned int *cuda_pind;

/* The mutex for accessing the cell pair list. */
__device__ int cuda_cell_mutex = 0;
__device__ int cuda_barrier = 0;
__device__ volatile int NAMD_barrier = 0;

/* The index of the next free cell pair. */
__device__ int cuda_pair_next = 0;

/* The list of cell pairs. */
__constant__ struct cellpair_cuda *cuda_pairs;
__device__ int *cuda_taboo;
#ifdef TASK_TIMERS
/*x = block y = type z = start w = end*/
__device__ int4 NAMD_timers[26*10000];
#endif

/* The list of tasks. */
__constant__ struct task_cuda *cuda_tasks;
__constant__ int cuda_nr_tasks = 0;

/* The per-SM task queues. */
__device__ struct queue_cuda cuda_queues[ cuda_maxqueues ];
__device__ struct queue_cuda cuda_sorts[ cuda_maxqueues ];
__constant__ int cuda_nrqueues;
__constant__ int cuda_queue_size;

/* Some constants. */
__constant__ float cuda_dt = 0.0f;
__constant__ float cuda_cutoff2 = 0.0f;
__constant__ float cuda_cutoff = 0.0f;
__constant__ float cuda_dscale = 0.0f;
__constant__ float cuda_maxdist = 0.0f;
__constant__ struct MxPotential **cuda_p;
__constant__ int cuda_maxtype = 0;
__constant__ struct MxPotentialCUDA *cuda_pots;
__constant__ int cuda_nr_pots;

/* Sortlists for the Verlet algorithm. */
__device__ unsigned int *cuda_sortlists = NULL;

/* Cell origins. */
__constant__ float *cuda_corig;

/* The potential parameters (hard-wired size for now). */
__constant__ float cuda_eps[ 100 ];
__constant__ float cuda_rmin[ 100 ];

/* Use a set of variables to communicate with the outside world. */
__device__ float cuda_fio[32];
__device__ int cuda_io[32];
__device__ int cuda_rcount = 0;

/* Potential energy. */
__device__ float cuda_epot = 0.0f, cuda_epot_out;

/* Timers. */
__device__ float cuda_timers[ tid_count ];

// Random number generators
__device__ hiprandState *cuda_rand_norm;

__global__ void cuda_init_rand_norm_device(hiprandState *rand_norm, int nr_rands, unsigned long long seed) {
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    int stride = blockDim.x * gridDim.x;
    while(tid < nr_rands) {
        hiprand_init(seed, tid, 0, &rand_norm[tid]);
        tid += stride;
    }
}

int engine_cuda_rand_norm_init(struct engine *e) {

    for(int did = 0; did < e->nr_devices; did++) {

        if(hipSetDevice(e->devices[did]) != hipSuccess)
            return cuda_error(engine_err_cuda);

        int nr_threads = e->nr_threads[did];
        int nr_blocks = e->nr_blocks[did];
        int nr_rands = nr_threads * nr_blocks;

        if(hipMalloc(&e->rand_norm_cuda[did], sizeof(hiprandState) * nr_rands) != hipSuccess)
            return cuda_error(engine_err_cuda);

        cuda_init_rand_norm_device<<<nr_blocks, nr_threads>>>((hiprandState *)e->rand_norm_cuda[did], nr_rands, e->rand_norm_seed_cuda);
        if(hipPeekAtLastError() != hipSuccess)
            return cuda_error(engine_err_cuda);

        if(hipMemcpyToSymbol(HIP_SYMBOL(cuda_rand_norm), &e->rand_norm_cuda[did], sizeof(void *), 0, hipMemcpyHostToDevice) != hipSuccess)
            return cuda_error(engine_err_cuda);

    }

    return engine_err_ok;

}

int engine_cuda_rand_norm_finalize(struct engine *e) {

    for(int did = 0; did < e->nr_devices; did++) {

        if(hipSetDevice(e->devices[did]) != hipSuccess)
            return cuda_error(engine_err_cuda);

        if(hipFree(e->rand_norm_cuda[did]) != hipSuccess)
            return cuda_error(engine_err_cuda);

    }
    
    return engine_err_ok;

}

/**
 * @brief Sets the random seed for the CUDA uniform number generators. 
 * 
 * @param e The #engine
 * @param seed The seed
 * @param onDevice A flag specifying whether the engine is current on the device
 * 
 * @return #engine_err_ok or < 0 on error (see #engine_err).
 */
extern "C" int engine_cuda_rand_norm_setSeed(struct engine *e, unsigned int seed, bool onDevice) {

    if(onDevice)
        if(engine_cuda_rand_norm_finalize(e) < 0)
            return cuda_error(engine_err_cuda);

    e->rand_norm_seed_cuda = seed;

    if(onDevice) {
        if(engine_cuda_rand_norm_init(e) < 0)
            return cuda_error(engine_err_cuda);

        for(int did = 0; did < e->nr_devices; did++) {

            if(hipSetDevice(e->devices[did]) != hipSuccess)
                return cuda_error(engine_err_cuda);

            if(hipDeviceSynchronize() != hipSuccess)
                return cuda_error(engine_err_cuda);

        }
    }

    return engine_err_ok;

}


/* Map sid to shift vectors. */
__constant__ float cuda_shiftn[13*3] = {
     5.773502691896258e-01 ,  5.773502691896258e-01 ,  5.773502691896258e-01 ,
     7.071067811865475e-01 ,  7.071067811865475e-01 ,  0.0                   ,
     5.773502691896258e-01 ,  5.773502691896258e-01 , -5.773502691896258e-01 ,
     7.071067811865475e-01 ,  0.0                   ,  7.071067811865475e-01 ,
     1.0                   ,  0.0                   ,  0.0                   ,
     7.071067811865475e-01 ,  0.0                   , -7.071067811865475e-01 ,
     5.773502691896258e-01 , -5.773502691896258e-01 ,  5.773502691896258e-01 ,
     7.071067811865475e-01 , -7.071067811865475e-01 ,  0.0                   ,
     5.773502691896258e-01 , -5.773502691896258e-01 , -5.773502691896258e-01 ,
     0.0                   ,  7.071067811865475e-01 ,  7.071067811865475e-01 ,
     0.0                   ,  1.0                   ,  0.0                   ,
     0.0                   ,  7.071067811865475e-01 , -7.071067811865475e-01 ,
     0.0                   ,  0.0                   ,  1.0                   ,
     };
__constant__ float cuda_shift[13*3] = {
     1.0 ,  1.0 ,  1.0 ,
     1.0 ,  1.0 ,  0.0 ,
     1.0 ,  1.0 , -1.0 ,
     1.0 ,  0.0 ,  1.0 ,
     1.0 ,  0.0 ,  0.0 ,
     1.0 ,  0.0 , -1.0 ,
     1.0 , -1.0 ,  1.0 ,
     1.0 , -1.0 ,  0.0 ,
     1.0 , -1.0 , -1.0 ,
     0.0 ,  1.0 ,  1.0 ,
     0.0 ,  1.0 ,  0.0 ,
     0.0 ,  1.0 , -1.0 ,
     0.0 ,  0.0 ,  1.0 ,
    };
    
/* The cell edge lengths and space dimensions. */
__constant__ float hip/hip_runtime.h[3];
__constant__ float cuda_dim[3];
    
    
/**
 * @brief Lock a device mutex.
 *
 * @param m The mutex.
 *
 * Loops until the mutex can be set. Note that only one thread
 * can do this at a time, so to synchronize blocks, only a single thread of
 * each block should call it.
 */

__device__ void cuda_mutex_lock ( int *m ) {
    TIMER_TIC
    while ( atomicCAS( m , 0 , 1 ) != 0 );
    TIMER_TOC( tid_mutex )
    }


/**
 * @brief Attempt to lock a device mutex.
 *
 * @param m The mutex.
 *
 * Try to grab the mutex. Note that only one thread
 * can do this at a time, so to synchronize blocks, only a single thread of
 * each block should call it.
 */

__device__ int cuda_mutex_trylock ( int *m ) {
    TIMER_TIC
    int res = atomicCAS( m , 0 , 1 ) == 0;
    TIMER_TOC( tid_mutex )
    return res;
    }


/**
 * @brief Lock a device mutex with an additional condition.
 *
 * @param m The mutex.
 * @param c the condition
 *
 * @return @c 1 if the mutex could be locked or zero if the condition @c c
 * was reached first.
 *
 * Loops until the mutex can be set or until @c *c is non-zero.
 * Note that only one thread
 * can do this at a time, so to synchronize blocks, only a single thread of
 * each block should call it.
 */

__device__ int cuda_mutex_lock_cond ( int *m , int *c ) {
    TIMER_TIC
    while ( atomicCAS( c , 0 , 0 ) == 0 )
        if ( atomicCAS( m , 0 , 1 ) == 0 ) {
            TIMER_TOC( tid_mutex )
            return 1;
            }
    TIMER_TOC( tid_mutex )
    return 0;
    }


/**
 * @brief Unlock a device mutex.
 *
 * @param m The mutex.
 *
 * Does not check if the mutex had been locked.
 */

__device__ void cuda_mutex_unlock ( int *m ) {
    TIMER_TIC
    atomicExch( m , 0 );
    TIMER_TOC( tid_mutex )
    }
    
    
/**
 * @brief Get a task ID from the given queue.
 *
 */
 
__device__ int cuda_queue_gettask ( struct queue_cuda *q ) {

    int ind, tid = -1;
    
    /* Don't even try... */
    if ( q->rec_count == q->count )
        return -1;

    /* Get the index of the next task. */
    ind = atomicAdd( &q->first , 1 );
        
    /* Wrap the index. */
    ind %= cuda_queue_size; 

    /* Loop until there is a valid task at that index. */
    while ( q->rec_count < q->count && ( tid = q->data[ind] ) < 0 );
    
    /* Scratch the task from the queue */
    if ( tid >= 0 )
        q->data[ind] = -1;

    /* Return the acquired task ID. */
    return tid;
    
    }


/**
 * @brief Put a task onto the given queue.
 *
 * @param tid The task ID to add to the end of the queue.
 */
 
__device__ void cuda_queue_puttask ( struct queue_cuda *q , int tid ) {

    int ind;

    /* Get the index of the next task. */
    ind = atomicAdd( &q->last , 1 ) % cuda_queue_size;
    
    /* Wait for the slot in the queue to be empty. */
    while ( q->data[ind] != -1 );

    /* Write the task back to the queue. */
    q->data[ind] = tid;
    
    }
    
    
/**
 * @brief Get the ID of the block's SM.
 */
 
__noinline__ __device__ uint get_smid ( void ) {
    uint ret;
    asm("mov.u32 %0, %smid;" : "=r"(ret) );
    return ret;
    }


/**
 * @brief Get a task from the given task queue.
 *
 * Picks tasks from the queue sequentially and checks if they
 * can be computed. If not, they are returned to the queue.
 *
 * This routine blocks until a valid task is picked up, or the
 * specified queue is empty.
 */
 
__device__ int runner_cuda_gettask ( struct queue_cuda *q , int steal ) {

    int tid = -1;
    int cid, cjd;
    
    TIMER_TIC
    
    /* Main loop. */
    while ( ( tid = cuda_queue_gettask( q ) ) >= 0 ) {
    
        /* If this task is not even free, don't even bother. */
        if ( !cuda_tasks[tid].wait ) {
    
            /* Dfferent options for different tasks. */
            if ( cuda_tasks[tid].type == task_type_sort ) {
            
                /* No locking needed. */
                break;
            
                }
            else if ( cuda_tasks[tid].type == task_type_self ) {
            
                /* Decode this task. */
                cid = cuda_tasks[tid].i;

                /* Lock down this task? */
                if ( cuda_mutex_trylock( &cuda_taboo[ cid ] ) )
                    break;
                        
                }
            else if ( cuda_tasks[tid].type == task_type_pair ) {
            
                /* Decode this task. */
                cid = cuda_tasks[tid].i;
                cjd = cuda_tasks[tid].j;

                /* Lock down this task? */
                if ( cuda_mutex_trylock( &cuda_taboo[ cid ] ) )
                    if ( cuda_mutex_trylock( &cuda_taboo[ cjd ] ) ) 
                        break;
                    else
                        cuda_mutex_unlock( &cuda_taboo[ cid ] );
                        
                }

            }
                
        /* Put this task back into the queue. */
        cuda_queue_puttask( q , tid );
    
        }
        
    /* Put this task into the recycling queue, if needed. */
    if ( tid >= 0 ) {
        if ( steal )
            atomicSub( (int *)&q->count , 1 );
        else
            q->rec_data[ atomicAdd( (int *)&q->rec_count , 1 ) ] = tid;
        }
        
    TIMER_TOC(tid_queue);
        
    /* Return whatever we got. */
    return tid;

    }

__device__ int runner_cuda_gettask_nolock ( struct queue_cuda *q , int steal ) {

    int tid = -1/*,cid,cjd*/;
    
    TIMER_TIC
    
    /* Main loop. */
    while ( ( tid = cuda_queue_gettask( q ) ) >= 0 ) {
    
        /* If this task is not even free, don't even bother. */
        if ( !cuda_tasks[tid].wait ) {
    
            break;

            }
        /*if( cuda_tasks[tid].type == task_type_pair )
        {
	       	cid = cuda_tasks[tid].i;
	       	cjd = cuda_tasks[tid].j;
	       	if(!( cuda_taboo[cid] || cuda_taboo[cjd] ))
	       		break;
       	}else{
       		break;
       	}*/
       	
                
        /* Put this task back into the queue. */
        cuda_queue_puttask( q , tid );
    
        }
        
    /* Put this task into the recycling queue, if needed. */
    if ( tid >= 0 ) {
        if ( steal )
            atomicSub( (int *)&q->count , 1 );
        else
            q->rec_data[ atomicAdd( (int *)&q->rec_count , 1 ) ] = tid;
        }
        
    TIMER_TOC(tid_queue);
        
    /* Return whatever we got. */
    return tid;

    }

__device__ inline void w_cubic_spline_cuda(float r2, float h, float *result) {
    float r = rsqrt(r2);
    float x = r/h;
    float y;
    
    if(x < 1.f) {
        float x2 = x * x;
        y = 1.f - (3.f / 2.f) * x2 + (3.f / 4.f) * x2 * x;
    }
    else if(x >= 1.f && x < 2.f) {
        float arg = 2.f - x;
        y = (1.f / 4.f) * arg * arg * arg;
    }
    else {
        y = 0.f;
    }
    
    *result = y / (M_PI * h * h * h);
}
    
/**
 * @brief Copy bulk memory in a strided way.
 *
 * @param dest Pointer to destination memory.
 * @param source Pointer to source memory.
 * @param count Number of bytes to copy, must be a multiple of sizeof(int).
 */
 
__device__ inline void cuda_memcpy ( void *dest , void *source , int count ) {

    int k;
    int *idest = (int *)dest, *isource = (int *)source;

    int threadID = threadIdx.x;
    
    TIMER_TIC
    
    /* Copy the data in chunks of sizeof(int). */
    for ( k = threadID ; k < count/sizeof(int) ; k += blockDim.x )
        idest[k] = isource[k];
        
    TIMER_TOC(tid_memcpy)
        
    }


    
__device__ inline void cuda_memcpy_old ( void *dest , void *source , int count ) {

    int k;
    volatile int *idest = (int *)dest, *isource = (int *)source;
    
    TIMER_TIC
    
    /* Copy the data in chunks of sizeof(int). */
    for ( k = 0 + threadIdx.x ; k < count/sizeof(int) ; k += cuda_frame )
        idest[k] = isource[k];
        
    TIMER_TOC(tid_memcpy)
        
    }


/**
 * @brief Sum two vectors in a strided way.
 *
 * @param a Pointer to destination memory.
 * @param b Pointer to source memory.
 * @param count Number of floats to sum.
 *
 * Computes @c a[k] += b[k] for k=1..count.
 */
 
__device__ inline void cuda_sum ( float *a , float *b , int count ) {

    int i, j, k, icount = count / cuda_frame / cuda_sum_chunk;
    float chunk[cuda_memcpy_chunk];
    int threadID = threadIdx.x;
    
    TIMER_TIC
    
    /* Copy the data in chunks of sizeof(int). */
    for ( k = 0 ; k < icount ; k += 1 ) {
        #pragma unroll
        for ( j = 0 ; j < cuda_sum_chunk ; j++ ) {
            i = (cuda_sum_chunk*k+j)*cuda_frame + threadID;
            chunk[j] = a[i] + b[i];
            }
        #pragma unroll
        for ( j = 0 ; j < cuda_sum_chunk ; j++ )
            a[ (cuda_sum_chunk*k+j)*cuda_frame + threadID ] = chunk[j];
        }
    for ( k = cuda_sum_chunk*cuda_frame*icount + threadID ; k < count ; k += cuda_frame )
        a[k] += b[k];
        
    TIMER_TOC(tid_update)
        
    }
    
    
    
    /**
 * @brief Sort the given data w.r.t. the lowest 16 bits in decending order.
 *
 * @param a The array to sort.
 * @param count The number of elements.
 */
 
__device__ inline void cuda_sort_descending ( unsigned int *a , int count ) {

    
    int i, j, k, threadID = threadIdx.x;
    int hi, lo, ind, jnd;
    unsigned int swap_i, swap_j;

    TIMER_TIC

    /* Sort using normalized bitonic sort. */
    for ( k = 1 ; k < count ; k *= 2 ) {
    
        /* First step. */
        for ( i = threadID ;  i < count ; i += blockDim.x ) {
            hi = i & ~(k-1); lo = i & (k-1);
            ind = i + hi; jnd = 2*(hi+k) - lo - 1;
            swap_i = ( jnd < count ) ? a[ind] : 0;
            swap_j = ( jnd < count ) ? a[jnd] : 0;
            if  ( ( swap_i & 0xffff ) < ( swap_j & 0xffff ) ) {
                a[ind] = swap_j;
                a[jnd] = swap_i;
                }
            }
            
        /* Let that last step sink in. */
            __syncthreads();
    
        /* Second step(s). */
        for ( j = k/2 ; j > 0 ; j /= 2 ) {
            for ( i = threadID ;  i < count ; i += blockDim.x ) {
                hi = i & ~(j-1);
                ind = i + hi; jnd = ind + j;
                swap_i = ( jnd < count ) ? a[ind] : 0;
                swap_j = ( jnd < count ) ? a[jnd] : 0;
                if  ( ( swap_i & 0xffff ) < ( swap_j & 0xffff ) ) {
                    a[ind] = swap_j;
                    a[jnd] = swap_i;
                    }
                }
                __syncthreads();
            }
            
        }
        
    TIMER_TOC(tid_sort)

        
    }

    


    
/**
 * @brief Sort the given data w.r.t. the lowest 16 bits in ascending order.
 *
 * @param a The array to sort.
 * @param count The number of elements.
 */
 
__device__ void cuda_sort_ascending ( unsigned int *a , int count ) {

    int i, j, k, threadID = threadIdx.x;
    int hi[2], lo[2], ind[2], jnd[2];
    unsigned int swap_i[2], swap_j[2];

    TIMER_TIC

    /* Sort using normalized bitonic sort. */
    for ( k = 1 ; k < count ; k *= 2 ) {
    
        /* First step. */
        for ( i = threadID ;  i < count ; i += 2*cuda_frame ) {
            hi[0] = i & ~(k-1); lo[0] = i & (k-1);
            hi[1] = (i + cuda_frame) & ~(k-1); lo[1] = (i + cuda_frame) & (k-1);
            ind[0] = i + hi[0]; jnd[0] = 2*(hi[0]+k) - lo[0] - 1;
            ind[1] = i + cuda_frame + hi[1]; jnd[1] = 2*(hi[1]+k) - lo[1] - 1;
            swap_i[0] = ( jnd[0] < count ) ? a[ind[0]] : 0;
            swap_i[1] = ( jnd[1] < count ) ? a[ind[1]] : 0;
            swap_j[0] = ( jnd[0] < count ) ? a[jnd[0]] : 0;
            swap_j[1] = ( jnd[1] < count ) ? a[jnd[1]] : 0;
            if  ( ( swap_i[0] & 0xffff ) > ( swap_j[0] & 0xffff ) ) {
                a[ind[0]] = swap_j[0];
                a[jnd[0]] = swap_i[0];
                }
            if  ( ( swap_i[1] & 0xffff ) > ( swap_j[1] & 0xffff ) ) {
                a[ind[1]] = swap_j[1];
                a[jnd[1]] = swap_i[1];
                }
            }
            
        /* Let that last step sink in. */
        // __threadfence_block();
    
        /* Second step(s). */
        for ( j = k/2 ; j > 0 ; j /= 2 ) {
            for ( i = threadID ;  i < count ; i += 2*cuda_frame ) {
                hi[0] = i & ~(j-1);
                hi[1] = (i + cuda_frame) & ~(j-1);
                ind[0] = i + hi[0]; jnd[0] = ind[0] + j;
                ind[1] = i + cuda_frame + hi[1]; jnd[1] = ind[1] + j;
                swap_i[0] = ( jnd[0] < count ) ? a[ind[0]] : 0;
                swap_i[1] = ( jnd[1] < count ) ? a[ind[1]] : 0;
                swap_j[0] = ( jnd[0] < count ) ? a[jnd[0]] : 0;
                swap_j[1] = ( jnd[1] < count ) ? a[jnd[1]] : 0;
                if  ( ( swap_i[0] & 0xffff ) > ( swap_j[0] & 0xffff ) ) {
                    a[ind[0]] = swap_j[0];
                    a[jnd[0]] = swap_i[0];
                    }
                if  ( ( swap_i[1] & 0xffff ) > ( swap_j[1] & 0xffff ) ) {
                    a[ind[1]] = swap_j[1];
                    a[jnd[1]] = swap_i[1];
                    }
                }
            // __threadfence_block();
            }
            
        }
        
    TIMER_TOC(tid_sort)
        
    }


/**
 * @brief Loads a potential onto a CUDA device
 * 
 * @param p The potential
 * 
 * @return The loaded potential, or NULL if failed
 */
MxPotential cuda_toDevice(MxPotential *p) {
    MxPotential p_d(*p);

    // Alloc and copy coefficients
    if(hipMalloc(&p_d.c, sizeof(FPTYPE) * (p->n + 1) * potential_chunk) != hipSuccess) {
        mx_error(E_FAIL, hipGetErrorString(hipPeekAtLastError()));
        return p_d;
    }
    if(hipMemcpy(p_d.c, p->c, sizeof(FPTYPE) * (p->n + 1) * potential_chunk, hipMemcpyHostToDevice) != hipSuccess) {
        mx_error(E_FAIL, hipGetErrorString(hipPeekAtLastError()));
        return p_d;
    }

    if(p->pca != NULL)
        *p_d.pca = cuda_toDevice(p->pca);
    if(p->pcb != NULL)
        *p_d.pcb = cuda_toDevice(p->pcb);

    return p_d;
}

/**
 * @brief Finalize a potential on a CUDA device
 * 
 * @param p The potential
 * 
 */
__device__ void cuda_finalize_onDevice(MxPotential &p) {
    hipFree(&p.c);
}

struct MxPotentialCUDA {

    // DPD coefficients alpha, gamma, sigma
    float3 dpd_cfs;

    // The potential
    MxPotential pot;

    __host__ __device__ 
    MxPotentialCUDA(MxPotential *p, bool toDevice=true) {
        if(toDevice) this->pot = cuda_toDevice(p);
        else this->pot = *p;

        if(p->kind == POTENTIAL_KIND_DPD) {
            DPDPotential *p_dpd = (DPDPotential*)p;
            this->dpd_cfs.x = p_dpd->alpha;
            this->dpd_cfs.y = p_dpd->gamma;
            this->dpd_cfs.z = p_dpd->sigma;
        }
    }
    
    __device__ 
    void finalize() {
        hipFree(&this->pot.c);
    }
};

struct MxParticleCUDA {
    int id;
    int typeId;
    float radius;
    float4 x;
    float4 v;

    __host__ __device__ 
    MxParticleCUDA() :
        id{-1}, 
        typeId{-1}
    {}

    __host__ __device__ 
    MxParticleCUDA(MxParticle *p) : 
        id{p->id}, 
        typeId{p->typeId}, 
        radius{p->radius}, 
        x{p->x[0], p->x[1], p->x[2], p->x[3]}, 
        v{p->v[0], p->v[1], p->v[2], p->v[3]}
    {}
};

MxParticleCUDA cuda_toDevice(MxParticle *p) {
    return MxParticleCUDA(p);
}

int cuda_fromDevice(MxParticleCUDA pc, MxParticle *p) {
    p->radius = pc.radius;
    
    p->x[0] = pc.x.x; p->x[1] = pc.x.y; p->x[2] = pc.x.z; p->x[3] = pc.x.w;
    p->v[0] = pc.v.x; p->v[1] = pc.v.y; p->v[2] = pc.v.z; p->v[3] = pc.v.w;

    return engine_err_ok;
}

/** 
 * @brief Evaluates the given potential at the given point (interpolated).
 *
 * @param p The #potential to be evaluated.
 * @param r2 The radius at which it is to be evaluated, squared.
 * @param e Pointer to a floating-point value in which to store the
 *      interaction energy.
 * @param f Pointer to a floating-point value in which to store the
 *      magnitude of the interaction force divided by r.
 *
 * Note that for efficiency reasons, this function does not check if any
 * of the parameters are @c NULL or if @c sqrt(r2) is within the interval
 * of the #potential @c p.
 */

__device__ inline void potential_eval_cuda ( struct MxPotential *p , float r2 , float *e , float *f ) {

    int ind, k;
    float x, ee, eff, *c, ir, r;
    
    TIMER_TIC
    
    /* Get r for the right type. */
    ir = rsqrtf(r2);
    r = r2*ir;
    
    /* compute the interval index */
    ind = fmaxf( 0.0f , p->alpha[0] + r * (p->alpha[1] + r * p->alpha[2]) );
    
    /* get the table offset */
    c = &(p->c[ind * potential_chunk]);
    
    /* adjust x to the interval */
    x = (r - c[0]) * c[1];
    
    /* compute the potential and its derivative */
    ee = c[2] * x + c[3];
    eff = c[2];
    #pragma unroll
    for ( k = 4 ; k < potential_chunk ; k++ ) {
        eff = eff * x + ee;
        ee = ee * x + c[k];
        }

    /* store the result */
    *e = ee; *f = eff * c[1] * ir;
        
    TIMER_TOC(tid_potential)
        
    }


/** 
 * @brief Evaluates the given potential at the given point (interpolated).
 *
 * @param p The #potential to be evaluated.
 * @param ri Radius of the ith particle. 
 * @param rj Radius of the jth particle. 
 * @param r2 The radius at which it is to be evaluated, squared.
 * @param e Pointer to a floating-point value in which to store the
 *      interaction energy.
 * @param f Pointer to a floating-point value in which to store the
 *      magnitude of the interaction force divided by r.
 *
 * Note that for efficiency reasons, this function does not check if any
 * of the parameters are @c NULL or if @c sqrt(r2) is within the interval
 * of the #potential @c p.
 */
__device__ inline void potential_eval_ex_cuda (struct MxPotential *p, float ri, float rj, float r2, float *e, float *f, bool *result) {

    int ind, k;
    float x, ee, eff, *c, ir, r;
    
    TIMER_TIC
    
    /* Get r for the right type. */
    ir = rsqrtf(r2);
    r = r2*ir;
    
    // cutoff min value, eval at lowest func interpolation.
    r = r < p->a ? p->a : r;
    
    if(p->flags & POTENTIAL_SCALED) {
        r = r / (ri + rj);
    }
    else if(p->flags & POTENTIAL_SHIFTED) {
        r = r - (ri + rj) + p->r0_plusone;
    }
    
    /* compute the interval index */
    ind = fmaxf( 0.0f , p->alpha[0] + r * (p->alpha[1] + r * p->alpha[2]) );

    if(r > p->b || ind > p->n) {
        *result = false;
        return;
    }
    
    /* get the table offset */
    c = &(p->c[ind * potential_chunk]);
    
    /* adjust x to the interval */
    x = (r - c[0]) * c[1];
    
    /* compute the potential and its derivative */
    ee = c[2] * x + c[3];
    eff = c[2];
    #pragma unroll
    for ( k = 4 ; k < potential_chunk ; k++ ) {
        eff = eff * x + ee;
        ee = ee * x + c[k];
        }

    /* store the result */
    *e = ee; *f = eff * c[1] * ir;

    *result = true;
        
    TIMER_TOC(tid_potential)
        
}


__device__ inline void dpd_eval_cuda(MxPotential pot, MxParticleCUDA pi, MxParticleCUDA pj, float3 dpd_cfs, float *dx, float r2, float *e, float *fi, float *fj, bool *result) {

    float delta = rsqrtf(cuda_dt);
    
    float ri = pi.radius;
    float rj = pj.radius;
    bool shifted = pot.flags & POTENTIAL_SHIFTED;
    
    float cutoff = shifted ? (pot.b + ri + rj) : pot.b;
    
    if(r2 > cutoff * cutoff) {
        *result = false;
        return;
    }
    
    float r = sqrtf(r2);
    
    if(r < pot.a) {
        *result = false;
        return;
    }
    
    // unit vector
    float3 unit_vec{dx[0] / r, dx[1] / r, dx[2] / r};
    
    float3 v{pi.v.x - pj.v.x, pi.v.y - pj.v.y, pi.v.z - pj.v.z};
    
    float shifted_r = shifted ? r - ri - rj : r;
    
    // conservative force
    float omega_c = shifted_r < 0.f ?  1.f : (1 - shifted_r / cutoff);
    
    float fc = dpd_cfs.x * omega_c;
    
    // dissapative force
    float omega_d = omega_c * omega_c;
    
    float fd = - dpd_cfs.y * omega_d * (unit_vec.x * v.x + unit_vec.y * v.y + unit_vec.z * v.z);
    
    float fr = dpd_cfs.z * omega_c * delta;
    
    float f = fc + fd + fr;
    
    fi[0] += f * unit_vec.x;
    fi[1] += f * unit_vec.y;
    fi[2] += f * unit_vec.z;
    fj[0] -= f * unit_vec.x;
    fj[1] -= f * unit_vec.y;
    fj[2] -= f * unit_vec.z;
    
    // TODO: correct energy
    *e = 0;

    *result = true;

}


// Underlying evaluation call; using templates to eliminate instructional overhead
template<uint32_t kind> 
__device__ inline void _potential_eval_super_ex_cuda(MxPotentialCUDA p_cuda, 
                                                    MxParticleCUDA pi, 
                                                    MxParticleCUDA pj, 
                                                    float *dx, 
                                                    float r2, 
                                                    float number_density, 
                                                    float *epot, 
                                                    float *fi, 
                                                    float *fj, 
                                                    bool *result) 
{
    float e;
    *result = false;
    float num_dens, _dx[3], _r2;
    auto flags = p_cuda.pot.flags;

    auto a = p_cuda.pot.a;
    
    if(flags & POTENTIAL_PERIODIC) {
        // Images don't contribute to density
        num_dens = 0.0;
        // Assuming elsewhere there's a corresponding potential in the opposite direction
        _dx[0] = dx[0] - p_cuda.pot.offset[0];
        _dx[1] = dx[1] - p_cuda.pot.offset[1];
        _dx[2] = dx[2] - p_cuda.pot.offset[2];
        _r2 = _dx[0]*_dx[0] + _dx[1]*_dx[1] + _dx[2]*_dx[2];
    }
    else {
        num_dens = number_density;
        _r2 = r2;
        for (int k = 0; k < 3; k++) _dx[k] = dx[k];
    }
    
    // if distance is less that potential min distance, define random
    // for repulsive force.
    if(_r2 < a * a) {
        int tid = threadIdx.x + blockDim.x * blockIdx.x;
        hiprandState *rand_norm = &cuda_rand_norm[tid];
        _dx[0] = hiprand_normal(rand_norm);
        _dx[1] = hiprand_normal(rand_norm);
        _dx[2] = hiprand_normal(rand_norm);
        float len = std::sqrt(_dx[0] * _dx[0] + _dx[1] * _dx[1] + _dx[2] * _dx[2]);
        _dx[0] = _dx[0] * p_cuda.pot.a / len;
        _dx[1] = _dx[1] * p_cuda.pot.a / len;
        _dx[2] = _dx[2] * p_cuda.pot.a / len;
        _r2 = p_cuda.pot.a * p_cuda.pot.a;
    }
    
    if(kind == POTENTIAL_KIND_DPD) {
        /* update the forces if part in range */
        dpd_eval_cuda(p_cuda.pot, pi, pj, p_cuda.dpd_cfs, _dx, _r2 , &e, fi, fj, result);
        
        if(*result) {
            
            // the number density is a union after the force 3-vector.
            fi[3] += num_dens;
            fj[3] += num_dens;
            
            /* tabulate the energy */
            *epot += e;
        }
    }
    else if(kind == POTENTIAL_KIND_BYPARTICLES) {
        // Currently not supported... ignoring
        *result = false;
    }
    else if(kind == POTENTIAL_KIND_COMBINATION) {
        if(flags & POTENTIAL_SUM) {
            bool resulta, resultb;
            potential_eval_super_ex_cuda(MxPotentialCUDA(p_cuda.pot.pca, false), pi, pj, _dx, _r2, num_dens, epot, fi, fj, &resulta);
            potential_eval_super_ex_cuda(MxPotentialCUDA(p_cuda.pot.pcb, false), pi, pj, _dx, _r2, num_dens, epot, fi, fj, &resultb);
            *result = resulta || resultb;
        }
    }
    else {
        float f;
    
        /* update the forces if part in range */
        potential_eval_ex_cuda(&p_cuda.pot, pi.radius, pj.radius, _r2 , &e , &f, result);
        if(*result) {
            
            for (int k = 0 ; k < 3 ; k++ ) {
                float w = f * _dx[k];
                fi[k] -= w;
                fj[k] += w;
            }
            
            // the number density is a union after the force 3-vector.
            fi[3] += num_dens;
            fj[3] += num_dens;
            
            /* tabulate the energy */
            *epot += e;
        }
    }
}

__device__ inline void potential_eval_super_ex_cuda(MxPotentialCUDA p_cuda, 
                                                    MxParticleCUDA pi, 
                                                    MxParticleCUDA pj, 
                                                    float *dx, 
                                                    float r2, 
                                                    float number_density, 
                                                    float *epot, 
                                                    float *fi, 
                                                    float *fj, 
                                                    bool *result) 
{
    uint32_t kind = p_cuda.pot.kind;

    if(kind == POTENTIAL_KIND_POTENTIAL) {
        _potential_eval_super_ex_cuda<POTENTIAL_KIND_POTENTIAL>(p_cuda, pi, pj, dx, r2, number_density, epot, fi, fj, result);
    }
    else if(kind == POTENTIAL_KIND_COMBINATION) {
        _potential_eval_super_ex_cuda<POTENTIAL_KIND_COMBINATION>(p_cuda, pi, pj, dx, r2, number_density, epot, fi, fj, result);
    }
    else if(kind == POTENTIAL_KIND_DPD) {
        _potential_eval_super_ex_cuda<POTENTIAL_KIND_DPD>(p_cuda, pi, pj, dx, r2, number_density, epot, fi, fj, result);
    }
}


/**
 * @brief Compute the pairwise interactions for the given pair on a CUDA device.
 *
 * @param icid Array of parts in the first cell.
 * @param count_i Number of parts in the first cell.
 * @param icjd Array of parts in the second cell.
 * @param count_j Number of parts in the second cell.
 * @param pshift A pointer to an array of three floating point values containing
 *      the vector separating the centers of @c cell_i and @c cell_j.
 * @param cid Part buffer in local memory.
 * @param cjd Part buffer in local memory.
 *
 * @sa #runner_dopair.
 */
__device__ void runner_dopair_unsorted_cuda ( MxParticleCUDA *parts_i , int count_i , MxParticleCUDA *parts_j , int count_j , float *forces_i , float *forces_j , float *shift , float *epot_global ) {
    int k, pid, pjd, ind, wrap_i, threadID;
    int pjoff;
    int pind;
    float epot = 0.0f, dx[3], pjf[4], r2;
    float ee = 0.0f;
    MxParticleCUDA pi, pj;
    float4 pjx;
    bool eval_result;
    float number_density;
    
    TIMER_TIC
    
    /* Get the size of the frame, i.e. the number of threads in this block. */
    threadID = threadIdx.x;
    
    /* Get the wraps. */
    wrap_i = (count_i < cuda_frame) ? cuda_frame : count_i;
    
    /* Make sure everybody is in the same place. */
    // __threadfence_block();

    /* Loop over the particles in cell_j, frame-wise. */
    for ( pjd = threadID ; pjd < count_j ; pjd += cuda_frame ) {
    
        /* Get a direct pointer on the pjdth part in cell_j. */
        pj = parts_j[ pjd ];
        pjoff = pj.typeId * cuda_maxtype;
        pjx = float4(pj.x);
        pjx.x += shift[0]; pjx.y += shift[1]; pjx.z += shift[2];
        pjf[0] = 0.0f; pjf[1] = 0.0f; pjf[2] = 0.0f; pjf[3] = 0.0f;
        
        /* Loop over the particles in cell_i. */
        for ( ind = 0 ; ind < wrap_i ; ind++ ) {
        
            /* Wrap the particle index correctly. */
            if ( ( pid = ind + threadID ) >= wrap_i )
                pid -= wrap_i;
            if ( pid < count_i ) {
            
                /* Get a handle on the wrapped particle pid in cell_i. */
                pi = parts_i[ pid ];

                /* Compute the radius between pi and pj. */
                r2 = 0.0f;
                dx[0] = pi.x.x - pjx.x; r2 += dx[0]*dx[0];
                dx[1] = pi.x.y - pjx.y; r2 += dx[1]*dx[1];
                dx[2] = pi.x.z - pjx.z; r2 += dx[2]*dx[2];
                pind = cuda_pind[pjoff + pi.typeId];

                if(pind == 0) {
                    continue;
                }

                /* Set the null potential if anything is bad. */
                if ( r2 < cuda_cutoff2 ) {

                    // atomicAdd( &cuda_rcount , 1 );
                    w_cubic_spline_cuda(r2, cuda_cutoff, &number_density);
                
                    /* Interact particles pi and pj. */
                    potential_eval_super_ex_cuda(cuda_pots[pind], pi, pj, dx, r2, number_density, &ee, &forces_i[4*pid], pjf, &eval_result);

                    /* Store the interaction force and energy. */
                    epot += ee;

                    /* Sync the shared memory values. */
                    // __threadfence_block();
                
                    } /* in range and potential. */

                } /* valid pid? */
        
            } /* loop over parts in cell_i. */
            
        /* Update the force on pj. */
        for ( k = 0 ; k < 4 ; k++ )
            forces_j[ 4*pjd + k ] += pjf[k];

        /* Sync the shared memory values. */
        // __threadfence_block();
            
        } /* loop over the particles in cell_j. */
        
    /* Store the potential energy. */
    *epot_global += epot;
        
    TIMER_TOC(tid_pair)
        
    }


/**
 * @brief Compute the pairwise interactions for the given pair on a CUDA device.
 *
 * @param icid Array of parts in the first cell.
 * @param count_i Number of parts in the first cell.
 * @param icjd Array of parts in the second cell.
 * @param count_j Number of parts in the second cell.
 * @param pshift A pointer to an array of three floating point values containing
 *      the vector separating the centers of @c cell_i and @c cell_j.
 * @param cid Part buffer in local memory.
 * @param cjd Part buffer in local memory.
 *
 * @sa #runner_dopair.
 */
__device__ void runner_dopair4_unsorted_cuda ( MxParticleCUDA *parts_i , int count_i , MxParticleCUDA *parts_j , int count_j , float *forces_i , float *forces_j , float *shift , float *epot_global ) {
    int k, pjd, ind, wrap_i, threadID;
    int pjoff;
    MxParticleCUDA pi[4], pj;
    float4 pjx;
    int4 pot, pid;
    char4 valid;
    float4 r2, ee;
    float epot = 0.0f, dx[12], pjf[4];
    bool eval_result;
    float4 number_density;
    
    TIMER_TIC
    
    /* Get the size of the frame, i.e. the number of threads in this block. */
    threadID = threadIdx.x;
    
    /* Get the wraps. */
    wrap_i = (count_i < cuda_frame) ? cuda_frame : count_i;
    
    /* Make sure everybody is in the same place. */
    // __threadfence_block();

    /* Loop over the particles in cell_j, frame-wise. */
    for ( pjd = threadID ; pjd < count_j ; pjd += cuda_frame ) {
    
        /* Get a direct pointer on the pjdth part in cell_j. */
        pj = parts_j[ pjd ];
        pjx = float4(pj.x);
        pjoff = pj.typeId * cuda_maxtype;
        pjx.x += shift[0]; pjx.y += shift[1]; pjx.z += shift[2];
        for ( k = 0 ; k < 4 ; k++ )
            pjf[k] = 0.0f;
        
        /* Loop over the particles in cell_i. */
        for ( ind = 0 ; ind < wrap_i ; ind += 4 ) {
        
            /* Wrap the particle index correctly. */
            if ( ( pid.x = ind + threadID ) >= wrap_i )
                pid.x -= wrap_i;
            if ( ( pid.y = ind + threadID + 1 ) >= wrap_i )
                pid.y -= wrap_i;
            if ( ( pid.z = ind + threadID + 2 ) >= wrap_i )
                pid.z -= wrap_i;
            if ( ( pid.w = ind + threadID + 3 ) >= wrap_i )
                pid.w -= wrap_i;
                
            /* Get the particle pointers. */
            pi[0] = ( valid.x = ( pid.x < count_i ) ) ? parts_i[ pid.x] : pj;
            pi[1] = ( valid.y = ( pid.y < count_i ) && ( ind + 1 < wrap_i ) ) ? parts_i[ pid.y ] : pj;
            pi[2] = ( valid.z = ( pid.z < count_i ) && ( ind + 2 < wrap_i ) ) ? parts_i[ pid.z ] : pj;
            pi[3] = ( valid.w = ( pid.w < count_i ) && ( ind + 3 < wrap_i ) ) ? parts_i[ pid.w ] : pj;
            
            /* Compute the pairwise distances. */
            r2 = make_float4( 0.0f );
            dx[0] = pjx.x - pi[0].x.x; r2.x += dx[0] * dx[0];
            dx[1] = pjx.y - pi[0].x.y; r2.x += dx[1] * dx[1];
            dx[2] = pjx.z - pi[0].x.z; r2.x += dx[2] * dx[2];
            dx[3] = pjx.x - pi[1].x.x; r2.y += dx[3] * dx[3];
            dx[4] = pjx.y - pi[1].x.y; r2.y += dx[4] * dx[4];
            dx[5] = pjx.z - pi[1].x.z; r2.y += dx[5] * dx[5];
            dx[6] = pjx.x - pi[2].x.x; r2.z += dx[6] * dx[6];
            dx[7] = pjx.y - pi[2].x.y; r2.z += dx[7] * dx[7];
            dx[8] = pjx.z - pi[2].x.z; r2.z += dx[8] * dx[8];
            dx[9] = pjx.x - pi[3].x.x; r2.w += dx[9] * dx[9];
            dx[10] = pjx.y - pi[3].x.y; r2.w += dx[10] * dx[10];
            dx[11] = pjx.z - pi[3].x.z; r2.w += dx[11] * dx[11];

                
            /* Get the potentials. */
            valid.x = ( valid.x && r2.x < cuda_cutoff2 );
            valid.y = ( valid.y && r2.y < cuda_cutoff2 );
            valid.z = ( valid.z && r2.z < cuda_cutoff2 );
            valid.w = ( valid.w && r2.w < cuda_cutoff2 );
            pot.x = valid.x ? cuda_pind[ pjoff + pi[0].typeId ] : 0;
            pot.y = valid.y ? cuda_pind[ pjoff + pi[1].typeId ] : 0;
            pot.z = valid.z ? cuda_pind[ pjoff + pi[2].typeId ] : 0;
            pot.w = valid.w ? cuda_pind[ pjoff + pi[3].typeId ] : 0;
            
            /* if ( pot.x != 0 )
                atomicAdd( &cuda_rcount , 1 );
            if ( pot.y != 0 )
                atomicAdd( &cuda_rcount , 1 );
            if ( pot.z != 0 )
                atomicAdd( &cuda_rcount , 1 );
            if ( pot.w != 0 )
                atomicAdd( &cuda_rcount , 1 ); */
            
            /* Update the forces. */
            if ( valid.x ) {
                w_cubic_spline_cuda(r2.x, cuda_cutoff, &number_density.x);
                potential_eval_super_ex_cuda(cuda_pots[pot.x], pi[0], pj, &dx[0], r2.x, number_density.x, &ee.x, pjf, &forces_i[4*pid.x], &eval_result);
                epot += ee.x;
                }
            // __threadfence_block();
            if ( valid.y ) {
                w_cubic_spline_cuda(r2.y, cuda_cutoff, &number_density.y);
                potential_eval_super_ex_cuda(cuda_pots[pot.y], pi[1], pj, &dx[3], r2.y, number_density.y, &ee.y, pjf, &forces_i[4*pid.y], &eval_result);
                epot += ee.y;
                }
            // __threadfence_block();
            if ( valid.z ) {
                w_cubic_spline_cuda(r2.z, cuda_cutoff, &number_density.z);
                potential_eval_super_ex_cuda(cuda_pots[pot.z], pi[2], pj, &dx[6], r2.z, number_density.z, &ee.z, pjf, &forces_i[4*pid.z], &eval_result);
                epot += ee.z;
                }
            // __threadfence_block();
            if ( valid.w ) {
                w_cubic_spline_cuda(r2.w, cuda_cutoff, &number_density.w);
                potential_eval_super_ex_cuda(cuda_pots[pot.w], pi[3], pj, &dx[9], r2.w, number_density.w, &ee.w, pjf, &forces_i[4*pid.w], &eval_result);
                epot += ee.w;
                }
            // __threadfence_block();
        
            } /* loop over parts in cell_i. */

        /* Update the force on pj. */
        for ( k = 0 ; k < 4 ; k++ )
            forces_j[ 4*pjd + k ] += pjf[k];

        /* Sync the shared memory values. */
        // __threadfence_block();
            
        } /* loop over the particles in cell_j. */
        
    /* Store the potential energy. */
    *epot_global += epot;
        
    TIMER_TOC(tid_pair)
        
    }


/**
 * @brief Compute the pairwise interactions for the given pair on a CUDA device.
 *
 * @param icid Array of parts in the first cell.
 * @param count_i Number of parts in the first cell.
 * @param icjd Array of parts in the second cell.
 * @param count_j Number of parts in the second cell.
 * @param pshift A pointer to an array of three floating point values containing
 *      the vector separating the centers of @c cell_i and @c cell_j.
 * @param cid Part buffer in local memory.
 * @param cjd Part buffer in local memory.
 *
 * @sa #runner_dopair.
 */
__device__ inline void runner_dosort_cuda ( MxParticleCUDA *parts_i , int count_i , unsigned int *sort_i , int sid ) {
    int k, threadID = threadIdx.x;
    MxParticleCUDA pi;
    // int4 spid;
    float nshift, shift[3], shiftn[3];
    
    TIMER_TIC
    
    /* Get the shift vector from the sid. */
    shift[0] = cuda_shift[ 3*sid + 0 ] * hip/hip_runtime.h[0];
    shift[1] = cuda_shift[ 3*sid + 1 ] * hip/hip_runtime.h[1];
    shift[2] = cuda_shift[ 3*sid + 2 ] * hip/hip_runtime.h[2];

    /* Pre-compute the inverse norm of the shift. */
    nshift = sqrtf( shift[0]*shift[0] + shift[1]*shift[1] + shift[2]*shift[2] );
    shiftn[0] = cuda_shiftn[ 3*sid + 0 ];
    shiftn[1] = cuda_shiftn[ 3*sid + 1 ];
    shiftn[2] = cuda_shiftn[ 3*sid + 2 ];



    /* Pack the parts into the sort arrays. */
    for ( k = threadID ; k < count_i ; k += blockDim.x ) {
        pi = parts_i[ k ];
        sort_i[k] = ( k << 16 ) | (unsigned int)( cuda_dscale * (nshift + pi.x.x*shiftn[0] + pi.x.y*shiftn[1] + pi.x.z*shiftn[2]) );
        }

    TIMER_TOC(tid_pack)
    __syncthreads();
    /* Sort using normalized bitonic sort. */
    cuda_sort_descending( sort_i , count_i );

    }
    
    
/**
 * @brief Compute the pairwise interactions for the given pair on a CUDA device.
 *
 * @param icid Array of parts in the first cell.
 * @param count_i Number of parts in the first cell.
 * @param icjd Array of parts in the second cell.
 * @param count_j Number of parts in the second cell.
 * @param pshift A pointer to an array of three floating point values containing
 *      the vector separating the centers of @c cell_i and @c cell_j.
 * @param cid Part buffer in local memory.
 * @param cjd Part buffer in local memory.
 *
 * @sa #runner_dopair.
 */
__device__ void runner_dopair_cuda ( MxParticleCUDA *parts_i , int count_i , MxParticleCUDA *parts_j , int count_j , float *forces_i , float *forces_j , unsigned int *sort_i , unsigned int *sort_j , float *shift , unsigned int dshift , float *epot_global ) {
    int k, pid, pjd, spid, spjd, pjdid, threadID, wrap, cj;
    int pioff;
    unsigned int dmaxdist;
    MxParticleCUDA pi, pj;
    float4 pix;
    int pind;
    float epot = 0.0f, r2, ee = 0.0f;
    float dx[3], pif[4];
    bool eval_result;
    float number_density;
    
    TIMER_TIC
    
    /* Get the size of the frame, i.e. the number of threads in this block. */
    threadID = threadIdx.x;
    
    /* Pre-compute the inverse norm of the shift. */
    dmaxdist = 2 + cuda_dscale * cuda_maxdist;
       

    /* Loop over the particles in cell_j, frame-wise. */
    cj = count_j;
    for ( pid = threadID ; pid < count_i ; pid += cuda_frame ) {
    
        /* Get the wrap. */
        while ( cj > 0 && ( sort_j[count_j-cj] & 0xffff ) + dshift - ( sort_i[pid & ~(cuda_frame - 1)] & 0xffff ) > dmaxdist )
            cj -= 1;
        if ( cj == 0 )
            break;
        else if ( cj < cuda_frame )
            wrap = max( cj , min( count_i - (pid & ~(cuda_frame - 1)) , cuda_frame ) );
        else
            wrap = cj;
            
        /* Get a direct pointer on the pjdth part in cell_j. */
        spid = sort_i[pid] >> 16;
        pi = parts_i[ spid ];
        pix = float4(pi.x);
        pioff = pi.typeId * cuda_maxtype;
        pix.x -= shift[0]; pix.y -= shift[1]; pix.z -= shift[2];
        pif[0] = 0.0f; pif[1] = 0.0f; pif[2] = 0.0f; pif[3] = 0.0f;
        
        /* Loop over the particles in cell_i. */

        for ( pjdid = 0 ; pjdid < wrap ; pjdid++ ) {
        
            /* Wrap the particle index correctly. */
            if ( ( pjd = pjdid + threadID ) >= wrap )
                pjd -= wrap;
            
            /* Do we have a pair? */
            if ( pjd < cj ) {
            
                /* Get a handle on the wrapped particle pid in cell_i. */
                spjd = sort_j[count_j-1-pjd] >> 16;
                pj = parts_j[ spjd ];

                /* Compute the radius between pi and pj. */
                r2 = 0.0f;
                dx[0] = pix.x - pj.x.x; r2 += dx[0]*dx[0];
                dx[1] = pix.y - pj.x.y; r2 += dx[1]*dx[1];
                dx[2] = pix.z - pj.x.z; r2 += dx[2]*dx[2];
                pind = cuda_pind[pioff + pj.typeId];

                if(pind == 0) {
                    continue;
                }
                    
                /* Set the null potential if anything is bad. */
	        
                 if ( r2 < cuda_cutoff2 ) {

                    /* printf( "runner_dopair_cuda[%i]: doing pair [%i,%i] with r=%i (d=%i).\n" ,
                        threadID , sort_i[pid].ind , sort_j[pjd].ind , (int)(sqrtf(r2)*1000.0) , (int)((sort_j[pjd].d - sort_i[pid].d)*1000) ); */

                    // atomicAdd( &cuda_pairs_done , 1 );
                    w_cubic_spline_cuda(r2, cuda_cutoff, &number_density);
                    
                    /* Interact particles pi and pj. */
                    potential_eval_super_ex_cuda(cuda_pots[pind], pi, pj, dx, r2, number_density, &ee, pif, &forces_j[4*spjd], &eval_result);

                    /* Store the interaction energy. */
                    epot += ee;

                    /* Sync the shared memory values. */
                    // __threadfence_block();
                
                     } /* in range and potential. */

                } /* do we have a pair? */
        
            } /* loop over parts in cell_i. */
            
        /* Update the force on pj. */
        for ( k = 0 ; k < 4 ; k++ )
            forces_i[ 4*spid + k ] += pif[k];
    
        /* Sync the shared memory values. */
        // __threadfence_block();
        
        } /* loop over the particles in cell_j. */
        
    /* Store the potential energy. */
    *epot_global += epot;
        
    TIMER_TOC(tid_pair)
    
    }


/**
 * @brief Compute the pairwise interactions for the given pair on a CUDA device.
 *
 * @param icid Array of parts in the first cell.
 * @param count_i Number of parts in the first cell.
 * @param icjd Array of parts in the second cell.
 * @param count_j Number of parts in the second cell.
 * @param pshift A pointer to an array of three floating point values containing
 *      the vector separating the centers of @c cell_i and @c cell_j.
 * @param cid Part buffer in local memory.
 * @param cjd Part buffer in local memory.
 *
 * @sa #runner_dopair.
 */
__device__  void runner_dopair_left_cuda ( MxParticleCUDA *parts_i , int count_i , MxParticleCUDA *parts_j , int count_j , float *forces_i , float *forces_j , unsigned int *sort_i , unsigned int *sort_j , float *shift , unsigned int dshift , float *epot_global ) {
    int k, pjd, spid, spjd, threadID;
    int pioff;
    unsigned int dmaxdist, di;
    MxParticleCUDA pi, pj;
    float4 pix;
    int pind;
    int i;
    float epot = 0.0f, r2, ee = 0.0f;
    float dx[3], pif[4], pjf[4];
    bool eval_result;
    float number_density;
    
    TIMER_TIC
    
    /* Get the size of the frame, i.e. the number of threads in this block. */
    threadID = threadIdx.x;
    
    /* Pre-compute the inverse norm of the shift. */
    dmaxdist = 2 + cuda_dscale * cuda_maxdist;

	/* nr_threads >= count_i */
    
    for ( i = threadID ; i < count_i ;  i += blockDim.x  ) {
    /*cj = count_j-1;
    while ( cj > 0 && ( sort_j[cj] & 0xffff ) + dshift  <= dmaxdist +( sort_i[threadID] & 0xffff ) )
            cj -= 1;*/
    di= sort_i[i]&0xffff;      
        /* Get a direct pointer on the pjdth part in cell_j. */
        spid = sort_i[i] >> 16;
        pi = parts_i[ spid ];
        pix = float4(pi.x);
        pioff = pi.typeId * cuda_maxtype;
        pix.x -= shift[0]; pix.y -= shift[1]; pix.z -= shift[2];
        pif[0] = 0.0f; pif[1] = 0.0f; pif[2] = 0.0f; pif[3] = 0.0f;
        /* Loop over the particles in cell_j. */
        for ( pjd = count_j-1 ; pjd >=0 && (sort_j[pjd]&0xffff)+dshift<=dmaxdist+di ; pjd-- ) {
                 
                /* Get a handle on the wrapped particle pid in cell_i. */

                spjd = sort_j[pjd] >> 16;
                pj = parts_j[ spjd ];

                /* Compute the radius between pi and pj. */
                r2 = 0.0f;
                dx[0] = pix.x - pj.x.x; r2 += dx[0]*dx[0];
                dx[1] = pix.y - pj.x.y; r2 += dx[1]*dx[1];
                dx[2] = pix.z - pj.x.z; r2 += dx[2]*dx[2];

                pind = cuda_pind[ pioff + pj.typeId ];
                if(pind == 0) {
                    continue;
                }
                    
                /* Set the null potential if anything is bad. */
                if ( r2 < cuda_cutoff2 ) {

                    /* printf( "runner_dopair_cuda[%i]: doing pair [%i,%i] with r=%i (d=%i).\n" ,
                        threadID , sort_i[pid].ind , sort_j[pjd].ind , (int)(sqrtf(r2)*1000.0) , (int)((sort_j[pjd].d - sort_i[pid].d)*1000) ); */

                    // atomicAdd( &cuda_pairs_done , 1 );
                    w_cubic_spline_cuda(r2, cuda_cutoff, &number_density);
                    
                    /* Interact particles pi and pj. */
                    potential_eval_super_ex_cuda(cuda_pots[pind], pi, pj, dx, r2, number_density, &ee, pif, pjf, &eval_result);

                    /* Store the interaction energy. */
                    epot += ee;

                    /* Sync the shared memory values. */
                    // __threadfence_block();
                
                    } /* in range and potential. */

            } /* loop over parts in cell_i. */
            
        /* Update the force on pj. */
        for ( k = 0 ; k < 4 ; k++ )
        	atomicAdd( &forces_i[ 4*spid + k], pif[k] );
            //forces_i[ 3*spid + k ] += pif[k];
    	
        /* Sync the shared memory values. */
        // __threadfence_block();
        
        } /* loop over the particles in cell_j. */
        
    /* Store the potential energy. */
    *epot_global += epot;
        
    TIMER_TOC(tid_pair)
    
    }




/**
 * @brief Compute the pairwise interactions for the given pair on a CUDA device.
 *
 * @param icid Array of parts in the first cell.
 * @param count_i Number of parts in the first cell.
 * @param icjd Array of parts in the second cell.
 * @param count_j Number of parts in the second cell.
 * @param pshift A pointer to an array of three floating point values containing
 *      the vector separating the centers of @c cell_i and @c cell_j.
 * @param cid Part buffer in local memory.
 * @param cjd Part buffer in local memory.
 *
 * @sa #runner_dopair.
 */
__device__ void runner_dopair_right_cuda ( MxParticleCUDA *parts_i , int count_i , MxParticleCUDA *parts_j , int count_j , float *forces_i , float *forces_j , unsigned int *sort_i , unsigned int *sort_j , float *shift , unsigned int dshift , float *epot_global ) {
    int k, pjd, spid, spjd, threadID;
    int pioff;
    unsigned int dmaxdist, dj;
    MxParticleCUDA pi, pj;
    float4 pix;
    int pind, i;
    float epot = 0.0f, r2, ee = 0.0f;
    float dx[3], pif[4], pjf[4];
    bool eval_result;
    float number_density;
    
    TIMER_TIC
    
    /* Get the size of the frame, i.e. the number of threads in this block. */
    threadID = threadIdx.x;
    
    /* Pre-compute the inverse norm of the shift. */
    dmaxdist = 2 + cuda_dscale * cuda_maxdist;
       

	/* nr_threads >= count_i */

    for ( i = threadID ; i < count_i ;  i += blockDim.x  ) {
    /*cj = 0;
    while ( cj < count_j && ( sort_i[threadID] & 0xffff ) + dshift  <= dmaxdist +( sort_j[cj] & 0xffff ) )
            cj += 1;*/
    dj = sort_i[i]&0xffff;      
        /* Get a direct pointer on the pjdth part in cell_j. */
        spid = sort_i[i] >> 16;
        pi = parts_i[ spid ];
        pix = float4(pi.x);
        pioff = pi.typeId * cuda_maxtype;
        pix.x += shift[0]; pix.y += shift[1]; pix.z += shift[2];
        pif[0] = 0.0f; pif[1] = 0.0f; pif[2] = 0.0f; pif[3] = 0.0f;
        
        /* Loop over the particles in cell_j. */
        for ( pjd = 0 ; pjd < count_j && dj+ dshift <= dmaxdist+(sort_j[pjd]&0xffff) ; pjd++ ) {
  				/*if((sort_i[threadID]&0xffff + dshift < dmaxdist))
		 			break;		               */
                /* Get a handle on the wrapped particle pid in cell_i. */
                spjd = sort_j[pjd] >> 16;
                pj = parts_j[ spjd ];

                /* Compute the radius between pi and pj. */
                r2 = 0.0f;
                dx[0] = pix.x - pj.x.x; r2 += dx[0]*dx[0];
                dx[1] = pix.y - pj.x.y; r2 += dx[1]*dx[1];
                dx[2] = pix.z - pj.x.z; r2 += dx[2]*dx[2];
                pind = cuda_pind[pioff + pj.typeId];

                if(pind == 0) {
                    continue;
                }
                    
                /* Set the null potential if anything is bad. */
                if ( r2 < cuda_cutoff2 ) {

                    /* printf( "runner_dopair_cuda[%i]: doing pair [%i,%i] with r=%i (d=%i).\n" ,
                        threadID , sort_i[pid].ind , sort_j[pjd].ind , (int)(sqrtf(r2)*1000.0) , (int)((sort_j[pjd].d - sort_i[pid].d)*1000) ); */

                    // atomicAdd( &cuda_pairs_done , 1 );
                    w_cubic_spline_cuda(r2, cuda_cutoff, &number_density);
                    
                    /* Interact particles pi and pj. */
                    potential_eval_super_ex_cuda(cuda_pots[pind], pi, pj, dx, r2, number_density, &ee, pif, pjf, &eval_result);

                    /* Store the interaction force and energy. */
                    epot += ee;

                    /* Sync the shared memory values. */
                    // __threadfence_block();
                
                    } /* in range and potential. */
            } /* loop over parts in cell_i. */
            
        /* Update the force on pj. */
        for ( k = 0 ; k < 4 ; k++ )
        	atomicAdd( &forces_i[ 4*spid + k] , pif[k]);
    		//forces_i[ 3*spid + k] += pif[k];
        /* Sync the shared memory values. */
        // __threadfence_block();
        
        } /* loop over the particles in cell_j. */
        
    /* Store the potential energy. */
    *epot_global += epot;
        
    TIMER_TOC(tid_pair)
    
}

/**
 * @brief Compute the self interactions for the given cell on a CUDA device.
 *
 * @param iparts Array of parts in this cell.
 * @param count Number of parts in the cell.
 * @param parts Part buffer in local memory.
 *
 * @sa #runner_dopair.
 */
__device__ void runner_doself_cuda ( MxParticleCUDA *parts , int count , float *forces , float *epot_global ) {
    int k, pid, threadID;
    int pjoff;
    MxParticleCUDA pi, pj;
    int pind, i;
    float epot = 0.0f, dx[3], pif[4], pjf[4], r2, ee;
    bool eval_result;
    float number_density;
    
    TIMER_TIC
    
    /* Get the size of the frame, i.e. the number of threads in this block. */
    threadID = threadIdx.x;
    
    /* Make sure everybody is in the same place. */
    // __threadfence_block();

    /* Loop over the particles in the cell, frame-wise. */
    for ( i = threadID ; i < count ;  i += blockDim.x  ) {
    
        /* Get a direct pointer on the pjdth part in cell_j. */
        pj = parts[ i ];
        pjoff = pj.typeId * cuda_maxtype;
        pjf[0] = 0.0f; pjf[1] = 0.0f; pjf[2] = 0.0f; pjf[3] = 0.0f;
            
        /* Loop over the particles in cell_i. */
        for ( pid = 0 ; pid < count ; pid++ ) {
        	if(i != pid ) {
            /* Get a handle on the wrapped particle pid in cell_i. */
            pi = parts[ pid ];

            /* Compute the radius between pi and pj. */
            r2 = 0.0f;
            dx[0] = pi.x.x - pj.x.x; r2 += dx[0]*dx[0];
            dx[1] = pi.x.y - pj.x.y; r2 += dx[1]*dx[1];
            dx[2] = pi.x.z - pj.x.z; r2 += dx[2]*dx[2];
            pind = cuda_pind[pjoff + pi.typeId];

            if(pind == 0) {
                continue;
            }

            /* Set the null potential if anything is bad. */
            if ( r2 < cuda_cutoff2 ) {
                w_cubic_spline_cuda(r2, cuda_cutoff, &number_density);

                /* Interact particles pi and pj. */
                potential_eval_super_ex_cuda(cuda_pots[pind], pi, pj, dx, r2, number_density, &ee, pif, pjf, &eval_result);

                /* Store the interaction force and energy. */
                epot += ee;

                /* Sync the shared memory values. */
                // __threadfence_block();
            
                } /* in range and potential. */
			}
            } /* loop over parts in cell_i. */
            
        /* Update the force on pj. */
        for ( k = 0 ; k < 4 ; k++ )
        	atomicAdd( &forces[ 4*i + k], pjf[k] );
			//forces[ 3*threadID + k] += pjf[k];
        /* Sync the shared memory values. */
        // __threadfence_block();

        } /* loop over the particles in cell_j. */
        
    /* Store the potential energy. */
    *epot_global += epot;
        
    TIMER_TOC(tid_self)
    
    }



/**
 * @brief Our very own memset for the particle forces as hipMemsetAsync requires
 *        a device switch when using streams on different devices.
 *
 */
 
__global__ void cuda_memset_float ( float *data , float val , int N ) {

    int k, tid = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = gridDim.x * blockDim.x;
    
    for ( k = tid ; k < N ; k += stride )
        data[k] = val;

    }


/** This set of defines and includes produces kernels with buffers for multiples
 *  of 32 particles up to 512 cuda_maxparts.
 */
 
#define cuda_nparts 32
    #include "runner_cuda_main.h"
#undef cuda_nparts

#define cuda_nparts 64
    #include "runner_cuda_main.h"
#undef cuda_nparts

#define cuda_nparts 96
    #include "runner_cuda_main.h"
#undef cuda_nparts

#define cuda_nparts 128
    #include "runner_cuda_main.h"
#undef cuda_nparts

#define cuda_nparts 160
    #include "runner_cuda_main.h"
#undef cuda_nparts

#define cuda_nparts 192
    #include "runner_cuda_main.h"
#undef cuda_nparts

#define cuda_nparts 224
    #include "runner_cuda_main.h"
#undef cuda_nparts

#define cuda_nparts 256
    #include "runner_cuda_main.h"
#undef cuda_nparts

#define cuda_nparts 288
    #include "runner_cuda_main.h"
#undef cuda_nparts

#define cuda_nparts 320
    #include "runner_cuda_main.h"
#undef cuda_nparts

#define cuda_nparts 352
    #include "runner_cuda_main.h"
#undef cuda_nparts

#define cuda_nparts 384
    #include "runner_cuda_main.h"
#undef cuda_nparts

#define cuda_nparts 416
    #include "runner_cuda_main.h"
#undef cuda_nparts

#define cuda_nparts 448
    #include "runner_cuda_main.h"
#undef cuda_nparts

#define cuda_nparts 480
    #include "runner_cuda_main.h"
#undef cuda_nparts

#define cuda_nparts 512
    #include "runner_cuda_main.h"
#undef cuda_nparts



/**
 * @brief Offload and compute the nonbonded interactions on a CUDA device.
 *
 * @param e The #engine.
 *
 * @return #engine_err_ok or < 0 on error (see #engine_err).
 */
 
extern "C" int engine_nonbond_cuda ( struct engine *e ) {

    int k, cid, did, pid, maxcount = 0;
    hipStream_t stream;
    hipEvent_t tic, toc_load, toc_run, toc_unload;
    float ms_load, ms_run, ms_unload;
    struct MxParticle *p;
    MxParticleCUDA *parts_cuda = (MxParticleCUDA *)e->parts_cuda_local, *buff4;
    struct space *s = &e->s;
    FPTYPE maxdist = s->cutoff + 2*s->maxdx;
    int *counts = e->counts_cuda_local[ 0 ], *inds = e->ind_cuda_local[ 0 ];
    float *forces_cuda[ engine_maxgpu ], epot[ engine_maxgpu ], *buff;
    #ifdef TIMERS
        float timers[ tid_count ];
        double icpms = 1000.0 / 1.4e9; 
    #endif
    
    /* Create the events. */
    if ( hipSetDevice( e->devices[e->nr_devices-1] ) ||
         hipEventCreate( &tic ) != hipSuccess ||
         hipEventCreate( &toc_load ) != hipSuccess ||
         hipEventCreate( &toc_run ) != hipSuccess ||
         hipEventCreate( &toc_unload ) != hipSuccess )
        return cuda_error(engine_err_cuda);
    
    /* Start the clock on the first stream. */
    if ( hipEventRecord( tic , (hipStream_t)e->streams[e->nr_devices-1] ) != hipSuccess )
        cuda_error(engine_err_cuda);
    
    /* Re-set timers */
    #ifdef TIMERS
        for ( int k = 0 ; k < tid_count ; k++ )
            timers[k] = 0.0f;
        for ( did = 0 ; did < e->nr_devices ; did++ )
            if ( hipMemcpyToSymbolAsync(HIP_SYMBOL( cuda_timers ), timers , sizeof(float) * tid_count , 0 , hipMemcpyHostToDevice , (hipStream_t)e->streams[did] ) != hipSuccess )
                return cuda_error(engine_err_cuda);
    #endif
    
    /* Loop over the devices and call the different kernels on each stream. */
    for ( did = 0 ; did < e->nr_devices ; did++ ) {
    
        /* Set the device ID. */
        if ( hipSetDevice( e->devices[did] ) != hipSuccess )
            return cuda_error(engine_err_cuda);

        /* Get the stream. */
        stream = (hipStream_t)e->streams[did];
    /* Load the particle data onto the device. */
    // tic = getticks();
    // if ( ( maxcount = engine_cuda_load_parts( e ) ) < 0 )
    //     return error(engine_err);
    // e->timers[ engine_timer_cuda_load ] += getticks() - tic;
    counts = e->counts_cuda_local[ did ];
    inds = e->ind_cuda_local[ did ];
    /* Clear the counts array. */
    bzero( counts , sizeof(int) * s->nr_cells );

    /* Load the counts. */
    for( maxcount = 0, k = 0; k < e->cells_cuda_nr[did] ; k++ )
	if( ( counts[e->cells_cuda_local[did][k]] = s->cells[e->cells_cuda_local[did][k]].count ) > maxcount )
	    maxcount = counts[ e->cells_cuda_local[did][k]];
/*    for ( maxcount = 0 , k = 0 ; k < s->nr_marked ; k++ )
        if ( ( counts[ s->cid_marked[k] ] = s->cells[ s->cid_marked[k] ].count ) > maxcount )
            maxcount = counts[ s->cid_marked[k] ];*/

    /* Raise maxcount to the next multiple of 32. */
    maxcount = ( maxcount + (cuda_frame - 1) ) & ~(cuda_frame - 1);
    // printf( "engine_cuda_load_parts: maxcount=%i.\n" , maxcount );

    /* Compute the indices. */
    inds[0] = 0;
    for ( k = 1 ; k < e->cells_cuda_nr[did] ; k++ )
        inds[k] = inds[k-1] + counts[k-1];

    /* Loop over the marked cells. */
    for ( k = 0 ; k < e->cells_cuda_nr[did] ; k++ ) {

        /* Get the cell id. */
        cid = e->cells_cuda_local[did][k];

        /* Copy the particle data to the device. */
        buff4 = (MxParticleCUDA *)&parts_cuda[ inds[cid] ];
        for ( pid = 0 ; pid < counts[cid] ; pid++ ) {
            buff4[ pid ] = cuda_toDevice(&s->cells[cid].parts[pid]);
            }

        }

	/* Start by setting the maxdist on the device. */
        if ( hipMemcpyToSymbolAsync(HIP_SYMBOL( cuda_maxdist ), &maxdist , sizeof(float) , 0 , hipMemcpyHostToDevice , stream ) != hipSuccess )
            return cuda_error(engine_err_cuda);

        /* Copy the counts onto the device. */
        if ( hipMemcpyAsync( e->counts_cuda[did] , counts , sizeof(int) * s->nr_cells , hipMemcpyHostToDevice , stream ) != hipSuccess )
            return cuda_error(engine_err_cuda);

        /* Copy the inds onto the device. */
        if ( hipMemcpyAsync( e->ind_cuda[did] , inds , sizeof(int) * s->nr_cells , hipMemcpyHostToDevice , stream ) != hipSuccess )
            return cuda_error(engine_err_cuda);

        /* Bind the particle positions. */
        if ( hipMemcpyAsync( e->parts_cuda[did] , parts_cuda , sizeof(MxParticleCUDA) * s->nr_parts , hipMemcpyHostToDevice , stream ) != hipSuccess )
            return cuda_error(engine_err_cuda);
    /* Start the clock. */
    // tic = getticks();
	}
    
    /* Lap the clock on the last stream. */
    if ( hipEventRecord( toc_load , (hipStream_t)e->streams[e->nr_devices-1] ) != hipSuccess )
        return cuda_error(engine_err_cuda);
    

        
        

	/* Loop over the devices and call the different kernels on each stream. */
    for ( did = 0 ; did < e->nr_devices ; did++ ) {

	/* Set the device ID. */
        if ( hipSetDevice( e->devices[did] ) != hipSuccess )
            return cuda_error(engine_err_cuda);

        /* Get the stream. */
        stream = (hipStream_t)e->streams[did];
        /* Clear the force array. */
        // if ( hipMemsetAsync( e->forces_cuda[did] , 0 , sizeof( float ) * 3 * s->nr_parts , stream ) != hipSuccess )
        //     return cuda_error(engine_err_cuda);
        cuda_memset_float <<<8,512,0,stream>>> ( e->forces_cuda[did] , 0.0f , 4 * s->nr_parts );

        dim3 nr_threads(e->nr_threads[did], 1);
        int nb = (s->nr_parts + nr_threads.x - 1) / nr_threads.x;
        dim3 nr_blocks(std::min(nb, e->nr_blocks[did]), 1);
            
        /* Start the appropriate kernel. */
        switch ( (maxcount + 31) / 32 ) {
            case 1:
                runner_run_cuda_32 <<<nr_blocks,nr_threads,0,stream>>> ( e->forces_cuda[did] , e->counts_cuda[did] , e->ind_cuda[did] , e->s.verlet_rebuild );
                break;
            case 2:
                runner_run_cuda_64 <<<nr_blocks,nr_threads,0,stream>>> ( e->forces_cuda[did] , e->counts_cuda[did] , e->ind_cuda[did] , e->s.verlet_rebuild );
                break;
            case 3:
                runner_run_cuda_96 <<<nr_blocks,nr_threads,0,stream>>> ( e->forces_cuda[did] , e->counts_cuda[did] , e->ind_cuda[did] , e->s.verlet_rebuild );
                break;
            case 4:
                runner_run_cuda_128 <<<nr_blocks,nr_threads,0,stream>>> ( e->forces_cuda[did] , e->counts_cuda[did] , e->ind_cuda[did] , e->s.verlet_rebuild );
                break;
            case 5:
                runner_run_cuda_160 <<<nr_blocks,nr_threads,0,stream>>> ( e->forces_cuda[did] , e->counts_cuda[did] , e->ind_cuda[did] , e->s.verlet_rebuild );
                break;
            case 6:
                runner_run_cuda_192 <<<nr_blocks,nr_threads,0,stream>>> ( e->forces_cuda[did] , e->counts_cuda[did] , e->ind_cuda[did] , e->s.verlet_rebuild );
                break;
            case 7:
                runner_run_cuda_224 <<<nr_blocks,nr_threads,0,stream>>> ( e->forces_cuda[did] , e->counts_cuda[did] , e->ind_cuda[did] , e->s.verlet_rebuild );
                break;
            case 8:
                runner_run_cuda_256 <<<nr_blocks,nr_threads,0,stream>>> ( e->forces_cuda[did] , e->counts_cuda[did] , e->ind_cuda[did] , e->s.verlet_rebuild );
                break;
            case 9:
                runner_run_cuda_288 <<<nr_blocks,nr_threads,0,stream>>> ( e->forces_cuda[did] , e->counts_cuda[did] , e->ind_cuda[did] , e->s.verlet_rebuild );
                break;
            case 10:
                runner_run_cuda_320 <<<nr_blocks,nr_threads,0,stream>>> ( e->forces_cuda[did] , e->counts_cuda[did] , e->ind_cuda[did] , e->s.verlet_rebuild );
                break;
            case 11:
                runner_run_cuda_352 <<<nr_blocks,nr_threads,0,stream>>> ( e->forces_cuda[did] , e->counts_cuda[did] , e->ind_cuda[did] , e->s.verlet_rebuild );
                break;
            case 12:
                runner_run_cuda_384 <<<nr_blocks,nr_threads,0,stream>>> ( e->forces_cuda[did] , e->counts_cuda[did] , e->ind_cuda[did] , e->s.verlet_rebuild );
                break;
            case 13:
                runner_run_cuda_416 <<<nr_blocks,nr_threads,0,stream>>> ( e->forces_cuda[did] , e->counts_cuda[did] , e->ind_cuda[did] , e->s.verlet_rebuild );
                break;
            case 14:
                runner_run_cuda_448 <<<nr_blocks,nr_threads,0,stream>>> ( e->forces_cuda[did] , e->counts_cuda[did] , e->ind_cuda[did] , e->s.verlet_rebuild );
                break;
            case 15:
                runner_run_cuda_480 <<<nr_blocks,nr_threads,0,stream>>> ( e->forces_cuda[did] , e->counts_cuda[did] , e->ind_cuda[did] , e->s.verlet_rebuild );
                break;
            case 16:
                runner_run_cuda_512 <<<nr_blocks,nr_threads,0,stream>>> ( e->forces_cuda[did] , e->counts_cuda[did] , e->ind_cuda[did] , e->s.verlet_rebuild );
                break;
            default:
                return error(engine_err_maxparts);
            }
}

	for( did = 0; did < e->nr_devices ; did ++ ) {
	
	/* Set the device ID. */
        if ( hipSetDevice( e->devices[did] ) != hipSuccess )
            return cuda_error(engine_err_cuda);
            
        /* Get the stream. */
        stream = (hipStream_t)e->streams[did];
        
        /* Get the forces from the device. */
        if ( ( forces_cuda[did] = (float *)malloc( sizeof(float) * 4 * s->nr_parts ) ) == NULL )
            return error(engine_err_malloc);
        if ( hipMemcpyAsync( forces_cuda[did] , e->forces_cuda[did] , sizeof(float) * 4 * s->nr_parts , hipMemcpyDeviceToHost , stream ) != hipSuccess )
            return cuda_error(engine_err_cuda);

        /* Get the potential energy. */
        if ( hipMemcpyFromSymbolAsync( &epot[did] , HIP_SYMBOL(cuda_epot_out ), sizeof(float) , 0 , hipMemcpyDeviceToHost , stream ) != hipSuccess )
            return cuda_error(engine_err_cuda);
        
        }
        
    // e->timers[ engine_timer_cuda_dopairs ] += getticks() - tic;
    
    /* Lap the clock on the last stream. */
    if ( hipEventRecord( toc_run , (hipStream_t)e->streams[e->nr_devices-1] ) != hipSuccess )
        return cuda_error(engine_err_cuda);
    
    /* Get and dump timers. */
    #ifdef TIMERS
        if ( hipMemcpyFromSymbolAsync( timers , HIP_SYMBOL(cuda_timers ), sizeof(float) * tid_count , 0 , hipMemcpyDeviceToHost , (hipStream_t)e->streams[0] ) != hipSuccess )
            return cuda_error(engine_err_cuda);
        printf( "engine_nonbond_cuda: timers = [ %.2f " , icpms * timers[0] );
        for ( int k = 1 ; k < tid_count ; k++ )
            printf( "%.2f " , icpms * timers[k] );
        printf( "] ms\n" );
    #endif

    #ifdef TASK_TIMERS
		int4 NAMD_timers_local[26*cuda_maxcells*3];
		if(hipMemcpyFromSymbol( NAMD_timers_local, HIP_SYMBOL(NAMD_timers), sizeof(int4)*26*cuda_maxcells*3 , 0 , hipMemcpyDeviceToHost) != hipSuccess )
			return cuda_error(engine_err_cuda);	
		for(int i = 0; i < e->s.nr_tasks ; i++)
		printf("Task: %i %i %i %i\n", NAMD_timers_local[i].x, NAMD_timers_local[i].y, NAMD_timers_local[i].z, NAMD_timers_local[i].w);

    #endif
    
    /* Check for any missed CUDA errors. */
    if ( hipPeekAtLastError() != hipSuccess )
        return cuda_error(engine_err_cuda);
        

    /* Loop over the devices. */
    for ( did = 0 ; did < e->nr_devices ; did++ ) {
    
        /* Get the stream. */
        stream = (hipStream_t)e->streams[did];
    
        /* Set the device ID. */
        // if ( hipSetDevice( e->devices[did] ) != hipSuccess )
        //     return cuda_error(engine_err_cuda);

        /* Wait for the chickens to come home to roost. */
        if ( hipStreamSynchronize( stream ) != hipSuccess )
            return cuda_error(engine_err_cuda);
    
        /* Get the potential energy. */
        e->s.epot += epot[did];
        
        /* Loop over the marked cells. */
        for ( k = 0 ; k < e->cells_cuda_nr[did] ; k++ ) {

            /* Get the cell id. */
            cid = e->cells_cuda_local[did][k];

            /* Copy the particle data from the device. */
            buff = &forces_cuda[did][ 4*e->ind_cuda_local[did][cid] ];
            for ( pid = 0 ; pid < s->cells[cid].count ; pid++ ) {
                p = &s->cells[cid].parts[pid];
                p->f[0] += buff[ 4*pid ];
                p->f[1] += buff[ 4*pid + 1 ];
                p->f[2] += buff[ 4*pid + 2 ];
                p->f[3] += buff[ 4*pid + 3 ];
                }

            }

        /* Deallocate the parts array and counts array. */
        free( forces_cuda[did] );
        
        }
        
    /* Check for any missed CUDA errors. */
    if ( hipPeekAtLastError() != hipSuccess )
        return cuda_error(engine_err_cuda);
        
    /* Stop the clock on the last stream. */
    if ( hipEventRecord( toc_unload , (hipStream_t)e->streams[e->nr_devices-1] ) != hipSuccess ||
         hipStreamSynchronize( (hipStream_t)e->streams[e->nr_devices-1] ) != hipSuccess )
        return cuda_error(engine_err_cuda);
    
    /* Check for any missed CUDA errors. */
    if ( hipPeekAtLastError() != hipSuccess )
        return cuda_error(engine_err_cuda);
        
    /* Store the timers. */
    if ( hipEventElapsedTime( &ms_load , tic , toc_load ) != hipSuccess ||
         hipEventElapsedTime( &ms_run , toc_load , toc_run ) != hipSuccess ||
         hipEventElapsedTime( &ms_unload , toc_run , toc_unload ) != hipSuccess )
        return cuda_error(engine_err_cuda);
    e->timers[ engine_timer_cuda_load ] += ms_load / 1000 * CPU_TPS;
    e->timers[ engine_timer_cuda_dopairs ] += ms_run / 1000 * CPU_TPS;
    e->timers[ engine_timer_cuda_unload ] += ms_unload / 1000 * CPU_TPS;
    
    /* Go away. */
    return engine_err_ok;
    
    }



/**
 * @brief Load the cell data onto the CUDA device.
 *
 * @param e The #engine.
 *
 * @return The maximum number of parts per cell or < 0
 *      on error (see #engine_err).
 */
 
extern "C" int engine_cuda_load_parts ( struct engine *e ) {
    
    int k, did, cid, pid, maxcount = 0;
    MxParticleCUDA *parts_cuda = (MxParticleCUDA *)e->parts_cuda_local, *buff;
    struct space *s = &e->s;
    FPTYPE maxdist = s->cutoff + 2*s->maxdx;
    int *counts = e->counts_cuda_local[0], *inds = e->ind_cuda_local[0];
    hipStream_t stream;
    
    /* Clear the counts array. */
    bzero( counts , sizeof(int) * s->nr_cells );

    /* Load the counts. */
    for ( maxcount = 0 , k = 0 ; k < s->nr_marked ; k++ )
        if ( ( counts[ s->cid_marked[k] ] = s->cells[ s->cid_marked[k] ].count ) > maxcount )
            maxcount = counts[ s->cid_marked[k] ];

    /* Raise maxcount to the next multiple of 32. */
    maxcount = ( maxcount + (cuda_frame - 1) ) & ~(cuda_frame - 1);
    // printf( "engine_cuda_load_parts: maxcount=%i.\n" , maxcount );

    /* Compute the indices. */
    inds[0] = 0;
    for ( k = 1 ; k < s->nr_cells ; k++ )
        inds[k] = inds[k-1] + counts[k-1];

    /* Loop over the marked cells. */
    for ( k = 0 ; k < s->nr_marked ; k++ ) {

        /* Get the cell id. */
        cid = s->cid_marked[k];

        /* Copy the particle data to the device. */
        buff = (MxParticleCUDA *)&parts_cuda[ inds[cid] ];
        for ( pid = 0 ; pid < counts[cid] ; pid++ ) {
            buff[ pid ] = cuda_toDevice(&s->cells[cid].parts[pid]);
            }

        }

    // printf( "engine_cuda_load_parts: packed %i cells with %i parts each (%i kB).\n" , s->nr_cells , maxcount , (sizeof(float4)*maxcount*s->nr_cells)/1024 );

    /* Loop over the devices. */
    for ( did = 0 ; did < e->nr_devices ; did++ ) {
    
        /* Set the device ID. */
        if ( hipSetDevice( e->devices[did] ) != hipSuccess )
            return cuda_error(engine_err_cuda);

        /* Get the stream. */
        stream = (hipStream_t)e->streams[did];
        
        /* Start by setting the maxdist on the device. */
        if ( hipMemcpyToSymbolAsync(HIP_SYMBOL( cuda_maxdist ), &maxdist , sizeof(float) , 0 , hipMemcpyHostToDevice , stream ) != hipSuccess )
            return cuda_error(engine_err_cuda);

        /* Copy the counts onto the device. */
        if ( hipMemcpyAsync( e->counts_cuda[did] , counts , sizeof(int) * s->nr_cells , hipMemcpyHostToDevice , stream ) != hipSuccess )
            return cuda_error(engine_err_cuda);

        /* Copy the inds onto the device. */
        if ( hipMemcpyAsync( e->ind_cuda[did] , inds , sizeof(int) * s->nr_cells , hipMemcpyHostToDevice , stream ) != hipSuccess )
            return cuda_error(engine_err_cuda);

        /* Bind the particle positions. */
        if ( hipMemcpyAsync( e->parts_cuda[did] , parts_cuda , sizeof(MxParticleCUDA) * s->nr_parts , hipMemcpyHostToDevice , stream ) != hipSuccess )
            return cuda_error(engine_err_cuda);

        /* Clear the force array. */
        if ( hipMemsetAsync( e->forces_cuda[did] , 0 , sizeof( float ) * 4 * s->nr_parts , stream ) != hipSuccess )
            return cuda_error(engine_err_cuda);
            
        }
    
    /* Our work is done here. */
    return maxcount;

    }
    
    

/**
 * @brief Load the cell data from the CUDA device.
 *
 * @param e The #engine.
 *
 * @return #engine_err_ok or < 0 on error (see #engine_err).
 */
 
extern "C" int engine_cuda_unload_parts ( struct engine *e ) {
    
    int k, did, cid, pid;
    struct MxParticle *p;
    float *forces_cuda[ engine_maxgpu ], *buff, epot[ engine_maxgpu ];
    struct space *s = &e->s;
    hipStream_t stream;
    
    /* Loop over the devices. */
    for ( did = 0 ; did < e->nr_devices ; did++ ) {
    
        /* Set the device ID. */
        if ( hipSetDevice( e->devices[did] ) != hipSuccess )
            return cuda_error(engine_err_cuda);

        /* Get the stream. */
        stream = (hipStream_t)e->streams[did];
    
        /* Get the forces from the device. */
        if ( ( forces_cuda[did] = (float *)malloc( sizeof(float) * 4 * s->nr_parts ) ) == NULL )
            return error(engine_err_malloc);
        if ( hipMemcpyAsync( forces_cuda[did] , e->forces_cuda[did] , sizeof(float) * 4 * s->nr_parts , hipMemcpyDeviceToHost , stream ) != hipSuccess )
            return cuda_error(engine_err_cuda);

        /* Get the potential energy. */
        if ( hipMemcpyFromSymbolAsync( &epot[did] , HIP_SYMBOL(cuda_epot_out ), sizeof(float) , 0 , hipMemcpyDeviceToHost , stream ) != hipSuccess )
            return cuda_error(engine_err_cuda);
        
        }

    /* Loop over the devices. */
    for ( did = 0 ; did < e->nr_devices ; did++ ) {
    
        /* Get the stream. */
        stream = (hipStream_t)e->streams[did];
    
        /* Set the device ID. */
        // if ( hipSetDevice( e->devices[did] ) != hipSuccess )
        //     return cuda_error(engine_err_cuda);

        /* Wait for the chickens to come home to roost. */
        if ( hipStreamSynchronize( stream ) != hipSuccess )
            return cuda_error(engine_err_cuda);
    
        /* Get the potential energy. */
        e->s.epot += epot[did];
        
        /* Loop over the marked cells. */
        for ( k = 0 ; k < s->nr_marked ; k++ ) {

            /* Get the cell id. */
            cid = s->cid_marked[k];

            /* Copy the particle data from the device. */
            buff = &forces_cuda[did][ 4*e->ind_cuda_local[did][cid] ];
            for ( pid = 0 ; pid < s->cells[cid].count ; pid++ ) {
                p = &s->cells[cid].parts[pid];
                p->f[0] += buff[ 4*pid ];
                p->f[1] += buff[ 4*pid + 1 ];
                p->f[2] += buff[ 4*pid + 2 ];
                p->f[3] += buff[ 4*pid + 3 ];
                }

            }

        /* Deallocate the parts array and counts array. */
        free( forces_cuda[did] );
        
        }
        
    /* Our work is done here. */
    return engine_err_ok;

    }

/**
 * @brief Load the queues onto the CUDA device.
 *
 * @param e The #engine.
 *
 * @return #engine_err_ok or < 0 on error (see #engine_err).
 */
 
int engine_cuda_queues_load ( struct engine *e ) {
    
    int did, nr_queues, qid, k, qsize, nr_tasks = e->s.nr_tasks;
    struct hipDeviceProp_t prop;
    int *data;
    struct queue_cuda queues[ cuda_maxqueues ];
    
    /* Loop over the devices. */
    for ( did = 0 ; did < e->nr_devices ; did++ ) {
    
        /* Set the device ID. */
        if ( hipSetDevice( e->devices[did] ) != hipSuccess )
            return cuda_error(engine_err_cuda);
            
        /* Get the device properties. */
        if ( hipGetDeviceProperties( &prop , e->devices[did] ) != hipSuccess )
            return cuda_error(engine_err_cuda);
            
        /* Get the number of SMs on the current device. */
        nr_queues = 1; // prop.multiProcessorCount;

        /* Get the local number of tasks. */
        nr_tasks = e->nrtasks_cuda[did];

        /* Set the size of each queue. */
        qsize = 3 * nr_tasks / min( nr_queues , e->nr_runners );
        if ( hipMemcpyToSymbol(HIP_SYMBOL( cuda_queue_size ), &qsize , sizeof(int) , 0 , hipMemcpyHostToDevice ) != hipSuccess )
            return cuda_error(engine_err_cuda);

        /* Allocate a temporary buffer for the queue data. */
        if ( ( data = (int *)malloc( sizeof(int) * qsize ) ) == NULL )
            return error(engine_err_malloc);

        /* Set the number of queues. */
        if ( hipMemcpyToSymbol(HIP_SYMBOL( cuda_nrqueues ), &nr_queues , sizeof(int) , 0 , hipMemcpyHostToDevice ) != hipSuccess )
            return cuda_error(engine_err_cuda);

        /* Init each queue separately. */
        for ( qid = 0 ; qid < nr_queues ; qid++ ) {

            /* Fill the data for this queue. */
            queues[qid].count = 0;
            for ( k = qid ; k < nr_tasks ; k += nr_queues )
                data[ queues[qid].count++ ] = k;
            for ( k = queues[qid].count ; k < qsize ; k++ )
                data[k] = -1;

            /* Allocate and copy the data. */
            if ( hipMalloc( &queues[qid].data , sizeof(int) * qsize ) != hipSuccess )
                return cuda_error(engine_err_cuda);
            if ( hipMemcpy( (void *)queues[qid].data , data , sizeof(int) * qsize , hipMemcpyHostToDevice ) != hipSuccess )
                return cuda_error(engine_err_cuda);

            /* Allocate and copy the recycling data. */
            for ( k = 0 ; k < queues[qid].count ; k++ )
                data[k] = -1;
            if ( hipMalloc( &queues[qid].rec_data , sizeof(int) * qsize ) != hipSuccess )
                return cuda_error(engine_err_cuda);
            if ( hipMemcpy( (void *)queues[qid].rec_data , data , sizeof(int) * qsize , hipMemcpyHostToDevice ) != hipSuccess )
                return cuda_error(engine_err_cuda);

            /* Set some other values. */
            queues[qid].first = 0;
            queues[qid].last = queues[qid].count;
            queues[qid].rec_count = 0;

            }

        /* Copy the queue structures to the device. */
        if ( hipMemcpyToSymbol(HIP_SYMBOL( cuda_queues ), queues , sizeof(struct queue_cuda) * nr_queues , 0 , hipMemcpyHostToDevice ) != hipSuccess )
            return cuda_error(engine_err_cuda);

        /* Wait so that we can re-use the local memory. */            
        if ( hipDeviceSynchronize() != hipSuccess )
            return cuda_error(engine_err_cuda);
            
        /* Clean up. */
        free( data );
        
        }
        
    /* Fade to grey. */
    return engine_err_ok;

    }


__global__ void engine_cuda_queues_finalize_device() {
    if(threadIdx.x != 0 || blockIdx.x != 0) {
        return;
    }

    for(int qid = 0; qid < cuda_nrqueues; qid++) {

        if(hipFree(&cuda_queues[qid].data) != hipSuccess) {
            printf("%s\n", "engine_cuda_queues_finalize_device failed (data)!");
            return;
        }

        if(hipFree(&cuda_queues[qid].rec_data) != hipSuccess) {
            printf("%s\n", "engine_cuda_queues_finalize_device failed (rec_data)!");
            return;
        }

    }

    if(hipFree(cuda_queues) != hipSuccess) {
        printf("%s\n", "engine_cuda_queues_finalize_device failed (cuda_queues)!");
        return;
    }
}

/**
 * @brief Close the run configuration on the CUDA device.
 *
 * @param e The #engine.
 *
 * @return #engine_err_ok or < 0 on error (see #engine_err).
 */
 
int engine_cuda_queues_finalize ( struct engine *e ) {

    /* Loop over the devices. */
    for(int did = 0 ; did < e->nr_devices ; did++ ) {
        
        // Set the device ID
        
        if ( hipSetDevice( e->devices[did] ) != hipSuccess )
            return cuda_error(engine_err_cuda);

        // Free queues
        
        engine_cuda_queues_finalize_device<<<1, 1>>>();

        if(hipDeviceSynchronize() != hipSuccess)
            return cuda_error(engine_err_cuda);

    }

    return engine_err_ok;
}

/**
 * @brief Load the potentials onto the CUDA device
 *
 * @param e The #engine.
 *
 * @return #engine_err_ok or < 0 on error (see #engine_err).
 */
extern "C" int engine_cuda_load_pots(struct engine *e) {
    int i, j, nr_pots, nr_coeffs, max_coeffs = 0;
    int did;
    int *pind = (int*)malloc(sizeof(int) * e->max_type * e->max_type);
    int nr_devices = e->nr_devices;
    struct MxPotential **pots = (MxPotential**)malloc(sizeof(MxPotential*) * e->nr_types * (e->nr_types + 1) / 2 + 1);

    /* Init the null potential. */
    if ( ( pots[0] = (struct MxPotential *)alloca( sizeof(struct MxPotential) ) ) == NULL )
        return error(engine_err_malloc);
    pots[0]->alpha[0] = pots[0]->alpha[1] = pots[0]->alpha[2] = pots[0]->alpha[3] = 0.0f;
    pots[0]->a = 0.0; pots[0]->b = FLT_MAX;
    pots[0]->flags = POTENTIAL_NONE;
    pots[0]->n = 0;
    if ( ( pots[0]->c = (FPTYPE *)alloca( sizeof(float) * potential_chunk ) ) == NULL )
        return error(engine_err_malloc);
    bzero( pots[0]->c , sizeof(float) * potential_chunk );
    nr_pots = 1; nr_coeffs = 1;
    
    /* Start by identifying the unique potentials in the engine. */
    for ( i = 0 ; i < e->max_type * e->max_type ; i++ ) {
    
        /* Skip if there is no potential or no parts of this type. */
        if ( e->p[i] == NULL )
            continue;
            
        /* Check this potential against previous potentials. */
        for ( j = 0 ; j < nr_pots && e->p[i] != pots[j] ; j++ );
        if ( j < nr_pots )
            continue;
            
        /* Store this potential and the number of coefficient entries it has. */
        pots[nr_pots] = e->p[i];
        nr_pots += 1;
        nr_coeffs += e->p[i]->n + 1;
        if ( e->p[i]->n + 1 > max_coeffs )
            max_coeffs = e->p[i]->n + 1;
    
    }

    /* Pack the potential matrix. */
    for ( i = 0 ; i < e->max_type * e->max_type ; i++ ) {
        if ( e->p[i] == NULL ) {
            pind[i] = 0;
            }
        else {
            for ( j = 0 ; j < nr_pots && pots[j] != e->p[i] ; j++ );
            pind[i] = j;
            }
        }

    // Pack the potentials
    MxPotentialCUDA *pots_cuda = (MxPotentialCUDA*)malloc(sizeof(MxPotentialCUDA) * nr_pots);
    for(i = 1; i < nr_pots; i++) {
        pots_cuda[i] = MxPotentialCUDA(pots[i]);
    }
    
    /* Store pind as a constant. */
    for ( did = 0 ; did < nr_devices ; did++ ) {
        if ( hipSetDevice( e->devices[did] ) != hipSuccess )
            return cuda_error(engine_err_cuda);
        if ( hipMalloc( &e->pind_cuda[did] , sizeof(unsigned int) * e->max_type * e->max_type ) != hipSuccess )
            return cuda_error(engine_err_cuda);
        if ( hipMemcpy( e->pind_cuda[did] , pind , sizeof(unsigned int) * e->max_type * e->max_type , hipMemcpyHostToDevice ) != hipSuccess )
            return cuda_error(engine_err_cuda);
        if ( hipMemcpyToSymbol(HIP_SYMBOL( cuda_pind ), &e->pind_cuda[did] , sizeof(void *) , 0 , hipMemcpyHostToDevice ) != hipSuccess )
            return cuda_error(engine_err_cuda);
    }
    free(pind);

    // Store the potentials
    for(did = 0; did < nr_devices; did++) {
        if(hipSetDevice(e->devices[did]) != hipSuccess)
            return cuda_error(engine_err_cuda);
        if(hipMalloc(&e->pots_cuda[did], sizeof(MxPotentialCUDA) * nr_pots) != hipSuccess)
            return cuda_error(engine_err_cuda);
        if(hipMemcpy(e->pots_cuda[did], pots_cuda, sizeof(MxPotentialCUDA) * nr_pots, hipMemcpyHostToDevice) != hipSuccess)
            return cuda_error(engine_err_cuda);
        if(hipMemcpyToSymbol(HIP_SYMBOL(cuda_pots), &e->pots_cuda[did], sizeof(void*), 0, hipMemcpyHostToDevice) != hipSuccess)
            return cuda_error(engine_err_cuda);
    }
    free(pots);
    free(pots_cuda);
    e->nr_pots_cuda = nr_pots;

    return engine_err_ok;
}

__global__ void unload_pots_device(int nr_pots) {
    int tid = threadIdx.x + gridDim.x * blockIdx.x;

    while(tid < nr_pots) {
        cuda_pots[tid].finalize();

        tid += gridDim.x;
    }
}

hipError_t finalize_pots_device(struct engine *e) {
    unload_pots_device<<<1, 1024>>>(e->nr_pots_cuda);
    return hipPeekAtLastError();
}

/**
 * @brief Unload the potentials on the CUDA device
 *
 * @param e The #engine.
 *
 * @return #engine_err_ok or < 0 on error (see #engine_err).
 */
extern "C" int engine_cuda_unload_pots(struct engine *e) {

    for(int did = 0; did < e->nr_devices; did++) {

        if(hipSetDevice(e->devices[did]) != hipSuccess)
            return cuda_error(engine_err_cuda);

        // Free the potentials.
        
        if(hipFree(e->pind_cuda[did]) != hipSuccess)
            return cuda_error(engine_err_cuda);

        // Free all potential members
        if(finalize_pots_device(e) != hipSuccess)
            return cuda_error(engine_err_cuda);
        
        if(hipFree((MxPotentialCUDA*)e->pots_cuda[did]) != hipSuccess)
            return cuda_error(engine_err_cuda);

    }

    e->nr_pots_cuda = 0;

    return engine_err_ok;
}

/**
 * @brief Refresh the potentials on the CUDA device. 
 * 
 * Can be safely called while on the CUDA device to reload all potential data from the engine. 
 * 
 * @param e The #engine
 * 
 * @return #engine_err_ok or < 0 on error (see #engine_err)
 */
extern "C" int engine_cuda_refresh_pots(struct engine *e) {
    
    if(engine_cuda_unload_pots(e) < 0)
        return error(engine_err);

    if(engine_cuda_load_pots(e) < 0)
        return error(engine_err);

    for(int did = 0; did < e->nr_devices; did++) {

        if(hipSetDevice(e->devices[did]) != hipSuccess)
            return cuda_error(engine_err_cuda);

        if(hipDeviceSynchronize() != hipSuccess)
            return cuda_error(engine_err_cuda);

    }

    return engine_err_ok;
}

/**
 * @brief Sets the number of particles on all current CUDA devices. 
 * 
 * @param nr_parts The current number of particles
 * 
 * @return #engine_err_ok or < 0 on error (see #engine_err).
 */
extern "C" int engine_cuda_update_nr_parts(struct engine *e) {

    /* Allocate the particle and force data. */
    for(int did = 0; did < e->nr_devices; did++) {
        if (hipSetDevice( e->devices[did] ) != hipSuccess )
            return cuda_error(engine_err_cuda);
        if (hipMemcpyToSymbol(HIP_SYMBOL(cuda_nr_parts), &e->s.nr_parts, sizeof(int), 0, hipMemcpyHostToDevice) != hipSuccess)
            return cuda_error(engine_err_cuda);
    }

    return engine_err_ok;

}

/**
 * @brief Allocates particle buffers. Must be called before running on a CUDA device. 
 * 
 * @param e The #engine
 * 
 * @return #engine_err_ok or < 0 on error (see #engine_err).
 */
int engine_cuda_allocate_particles(struct engine *e) {

    /* Allocate the particle buffer. */
    if((e->parts_cuda_local = (MxParticleCUDA*)malloc(sizeof(MxParticleCUDA) * e->s.size_parts)) == NULL)
        return error(engine_err_malloc);

    /* Allocate the particle and force data. */
    for(int did = 0; did < e->nr_devices; did++) {
        if (hipSetDevice( e->devices[did] ) != hipSuccess )
            return cuda_error(engine_err_cuda);
        if (hipMemcpyToSymbol(HIP_SYMBOL(cuda_nr_parts), &e->s.nr_parts, sizeof(int), 0, hipMemcpyHostToDevice) != hipSuccess)
            return cuda_error(engine_err_cuda);
        if (hipMemcpyToSymbol(HIP_SYMBOL(cuda_size_parts), &e->s.size_parts, sizeof(int), 0, hipMemcpyHostToDevice) != hipSuccess)
            return cuda_error(engine_err_cuda);
        if (hipMalloc(&e->parts_cuda[did], sizeof(MxParticleCUDA) * e->s.size_parts) != hipSuccess)
            return cuda_error(engine_err_cuda);
        if (hipMemcpyToSymbol(HIP_SYMBOL(cuda_parts), &e->parts_cuda[did], sizeof(void *), 0, hipMemcpyHostToDevice) != hipSuccess)
            return cuda_error(engine_err_cuda);
        if (hipMalloc(&e->forces_cuda[did], sizeof(float) * 4 * e->s.size_parts) != hipSuccess)
            return cuda_error(engine_err_cuda);
    }

    return engine_err_ok;
}


/**
 * @brief Closes particle buffers. 
 * 
 * @param e The #engine
 * 
 * @return #engine_err_ok or < 0 on error (see #engine_err).
 */
int engine_cuda_finalize_particles(struct engine *e) {

    for(int did = 0; did < e->nr_devices; did++) {

        if(hipSetDevice(e->devices[did]) != hipSuccess)
            return cuda_error(engine_err_cuda);

        // Free the particle and force data

        if(hipFree(e->parts_cuda[did]) != hipSuccess)
            return cuda_error(engine_err_cuda);

        if(hipFree(e->forces_cuda[did]) != hipSuccess)
            return cuda_error(engine_err_cuda);

    }

    // Free the particle buffer

    free(e->parts_cuda_local);

    return engine_err_ok;
}


/**
 * @brief Refreshes particle buffers. Can be safely used to resize buffers while running on CUDA device. 
 * 
 * @param e The #engine
 * 
 * @return #engine_err_ok or < 0 on error (see #engine_err).
 */
extern "C" int engine_cuda_refresh_particles(struct engine *e) {
    
    if(engine_cuda_finalize_particles(e) < 0)
        return cuda_error(engine_err_cuda);

    if(engine_cuda_allocate_particles(e) < 0)
        return cuda_error(engine_err_cuda);

    for(int did = 0; did < e->nr_devices; did++) {

        if(hipSetDevice(e->devices[did]) != hipSuccess)
            return cuda_error(engine_err_cuda);

        if(hipDeviceSynchronize() != hipSuccess)
            return cuda_error(engine_err_cuda);

    }

    return engine_err_ok;
}


/**
 * @brief Load the potentials and cell pairs onto the CUDA device.
 *
 * @param e The #engine.
 *
 * @return #engine_err_ok or < 0 on error (see #engine_err).
 */
 
extern "C" int engine_cuda_load ( struct engine *e ) {

    int i, k, nr_tasks, c1 ,c2;
    int did, *cellsorts;
    struct space *s = &e->s;
    int nr_devices = e->nr_devices;
    struct task_cuda *tasks_cuda, *tc, *ts;
    struct task *t;
    float dt = e->dt, cutoff = e->s.cutoff, cutoff2 = e->s.cutoff2, dscale; //, buff[ e->nr_types ];
    float h[3], dim[3], *corig;
    void *dummy[ engine_maxgpu ];

    /*Split the space over the available GPUs*/
    engine_split_gpu( e , nr_devices , engine_split_GPU  );
    
    /* Copy the cell edge lengths to the device. */
    h[0] = s->h[0]*s->span[0];
    h[1] = s->h[1]*s->span[1];
    h[2] = s->h[2]*s->span[2];
    dim[0] = s->dim[0]; dim[1] = s->dim[1]; dim[2] = s->dim[2];
    for ( did = 0 ; did < nr_devices ; did++ ) {
        if ( hipSetDevice( e->devices[did] ) != hipSuccess )
            return cuda_error(engine_err_cuda);
        if ( hipMemcpyToSymbol(HIP_SYMBOL( hip/hip_runtime.h ), h , sizeof(float) * 3 , 0 , hipMemcpyHostToDevice ) != hipSuccess )
            return cuda_error(engine_err_cuda);
        if ( hipMemcpyToSymbol(HIP_SYMBOL( cuda_dim ), dim , sizeof(float) * 3 , 0 , hipMemcpyHostToDevice ) != hipSuccess )
            return cuda_error(engine_err_cuda);
        }
        
    /* Copy the cell origins to the device. */
    if ( ( corig = (float *)malloc( sizeof(float) * s->nr_cells * 3 ) ) == NULL )
        return error(engine_err_malloc);
    for ( i = 0 ; i < s->nr_cells ; i++ ) {
        corig[ 3*i + 0 ] = s->cells[i].origin[0];
        corig[ 3*i + 1 ] = s->cells[i].origin[1];
        corig[ 3*i + 2 ] = s->cells[i].origin[2];
        }
    for ( did = 0 ; did < nr_devices ; did++ ) {
        if ( hipSetDevice( e->devices[did] ) != hipSuccess )
            return cuda_error(engine_err_cuda);
        if ( hipMalloc( &dummy[did] , sizeof(float) * s->nr_cells * 3 ) != hipSuccess )
            return cuda_error(engine_err_cuda);
        if ( hipMemcpy( dummy[did] , corig , sizeof(float) * s->nr_cells * 3 , hipMemcpyHostToDevice ) != hipSuccess )
            return cuda_error(engine_err_cuda);
        if ( hipMemcpyToSymbol(HIP_SYMBOL( cuda_corig ), &dummy[did] , sizeof(void *) , 0 , hipMemcpyHostToDevice ) != hipSuccess )
            return cuda_error(engine_err_cuda);
        }
    free( corig );
        
    /* Set the constant pointer to the null potential and other useful values. */
    dscale = ((float)SHRT_MAX) / ( 3.0 * sqrt( s->h[0]*s->h[0]*s->span[0]*s->span[0] + s->h[1]*s->h[1]*s->span[1]*s->span[1] + s->h[2]*s->h[2]*s->span[2]*s->span[2] ) );
    for ( did = 0 ;did < nr_devices ; did++ ) {
        if ( hipSetDevice( e->devices[did] ) != hipSuccess )
            return cuda_error(engine_err_cuda);
        if ( hipMemcpyToSymbol(HIP_SYMBOL( cuda_dt ), &dt , sizeof(float) , 0 , hipMemcpyHostToDevice ) != hipSuccess )
            return cuda_error(engine_err_cuda);
        if ( hipMemcpyToSymbol(HIP_SYMBOL( cuda_cutoff ), &cutoff , sizeof(float) , 0 , hipMemcpyHostToDevice ) != hipSuccess )
            return cuda_error(engine_err_cuda);
        if ( hipMemcpyToSymbol(HIP_SYMBOL( cuda_cutoff2 ), &cutoff2 , sizeof(float) , 0 , hipMemcpyHostToDevice ) != hipSuccess )
            return cuda_error(engine_err_cuda);
        if ( hipMemcpyToSymbol(HIP_SYMBOL( cuda_maxdist ), &cutoff , sizeof(float) , 0 , hipMemcpyHostToDevice ) != hipSuccess )
            return cuda_error(engine_err_cuda);
        if ( hipMemcpyToSymbol(HIP_SYMBOL( cuda_maxtype ), &(e->max_type) , sizeof(int) , 0 , hipMemcpyHostToDevice ) != hipSuccess )
            return cuda_error(engine_err_cuda);
        if ( hipMemcpyToSymbol(HIP_SYMBOL( cuda_dscale ), &dscale , sizeof(float) , 0 , hipMemcpyHostToDevice ) != hipSuccess )
            return cuda_error(engine_err_cuda);
        if ( hipMemcpyToSymbol(HIP_SYMBOL( cuda_nr_cells ), &(s->nr_cells) , sizeof(int) , 0 , hipMemcpyHostToDevice ) != hipSuccess )
            return cuda_error(engine_err_cuda);
        }
        
    /* Allocate and fill the task list. */
    if ( ( tasks_cuda = (struct task_cuda *)malloc( sizeof(struct task_cuda) * s->nr_tasks ) ) == NULL )
        return error(engine_err_malloc);
    if ( ( cellsorts = (int *)malloc( sizeof(int) * s->nr_tasks ) ) == NULL )
        return error(engine_err_malloc);
    for ( did = 0 ;did < nr_devices ; did++ ) {
	if( (e->cells_cuda_local[did] = (int *)malloc( sizeof(int) * s->nr_cells ) ) == NULL)
	    return error(engine_err_malloc);
        e->cells_cuda_nr[did]=0;
        if ( hipSetDevice( e->devices[did] ) != hipSuccess )
            return cuda_error(engine_err_cuda);
        /* Select the tasks for each device ID. */  
        for ( nr_tasks = 0 , i = 0 ; i < s->nr_tasks ; i++ ) {
            
            /* Get local pointers. */
            t = &s->tasks[i];
            tc = &tasks_cuda[nr_tasks];
	    
            /* Skip pairs and self with wrong cid, keep all sorts. */
            if ( ( t->type == task_type_pair && e->s.cells[t->i].GPUID != did  /*t->i % nr_devices != did */) ||
                 ( t->type == task_type_self && e->s.cells[t->i].GPUID != did /*e->s.cells[t->i].loc[1] < e->s.cdim[1] / e->nr_devices * (did + 1) && e->s.cells[t->i].loc[1] >= e->s.cdim[1] / e->nr_devices * did t->i % e->nr_devices != did*/ ) )
                continue;
            
            /* Copy the data. */
            tc->type = t->type;
            tc->subtype = t->subtype;
            tc->wait = 0;
            tc->flags = t->flags;
            tc->i = t->i;
            tc->j = t->j;
            tc->nr_unlock = 0;
            
            /* Remember which task sorts which cell. */
            if ( t->type == task_type_sort ) {
                tc->flags = 0;
                cellsorts[ t->i ] = nr_tasks;
                }

	    /*Add the cell to list of cells for this GPU if needed*/
  	    c1=1; c2=1;
	    for(int i = 0; i < e->cells_cuda_nr[did] ; i++ )
	    {
	    /* Check cell is valid */
		if(t->i < 0 || t->i == e->cells_cuda_local[did][i])
		    c1 = 0;
		if(t->j < 0 || t->j == e->cells_cuda_local[did][i])
		    c2 = 0;
   	    }
	    if( c1 )
		e->cells_cuda_local[did][e->cells_cuda_nr[did]++] = t->i;
	    if( c2 )
		e->cells_cuda_local[did][e->cells_cuda_nr[did]++] = t->j;	                
            /* Add one task. */
            nr_tasks += 1;
		
            }

        /* Link each pair task to its sorts. */
        for ( i = 0 ; i < nr_tasks ; i++ ) {
            tc = &tasks_cuda[i];
	
            if ( tc->type == task_type_pair ) {
                ts = &tasks_cuda[ cellsorts[ tc->i ] ];
                ts->flags |= (1 << tc->flags);
                ts->unlock[ ts->nr_unlock ] = i;
                ts->nr_unlock += 1;
                ts = &tasks_cuda[ cellsorts[ tc->j ] ];
                ts->flags |= (1 << tc->flags);
                ts->unlock[ ts->nr_unlock ] = i;
                ts->nr_unlock += 1;
                }
            }
        
        /* Set the waits. */
        for ( i = 0 ; i < nr_tasks ; i++ )
            for ( k = 0 ; k < tasks_cuda[i].nr_unlock ; k++ )
                tasks_cuda[ tasks_cuda[i].unlock[k] ].wait += 1;

        /* Allocate and fill the tasks list on the device. */
        if ( hipMemcpyToSymbol(HIP_SYMBOL( cuda_nr_tasks ), &nr_tasks , sizeof(int) , 0 , hipMemcpyHostToDevice ) != hipSuccess )
            return cuda_error(engine_err_cuda);
        if ( hipMalloc( &dummy[did] , sizeof(struct task_cuda) * s->nr_tasks ) != hipSuccess )
            return cuda_error(engine_err_cuda);
        if ( hipMemcpy( dummy[did] , tasks_cuda , sizeof(struct task_cuda) * s->nr_tasks , hipMemcpyHostToDevice ) != hipSuccess )
            return cuda_error(engine_err_cuda);
        if ( hipMemcpyToSymbol(HIP_SYMBOL( cuda_tasks ), &dummy[did] , sizeof(struct task_cuda *) , 0 , hipMemcpyHostToDevice ) != hipSuccess )
            return cuda_error(engine_err_cuda);
            
        /* Remember the number of tasks. */
        e->nrtasks_cuda[did] = nr_tasks;
            
        }
    
	/* Clean up */
    free( tasks_cuda );
    free( cellsorts );
        
    /* Allocate the sortlists locally and on the device if needed. */
    for ( did = 0 ;did < nr_devices ; did++ ) {
        if ( hipSetDevice( e->devices[did] ) != hipSuccess )
            return cuda_error(engine_err_cuda);
        if ( hipMalloc( &e->sortlists_cuda[did] , sizeof(unsigned int) * s->nr_parts * 13 ) != hipSuccess )
            return cuda_error(engine_err_cuda);
        if ( hipMemcpyToSymbol(HIP_SYMBOL( cuda_sortlists ), &e->sortlists_cuda[did] , sizeof(void *) , 0 , hipMemcpyHostToDevice ) != hipSuccess )
            return cuda_error(engine_err_cuda);
        }


    for ( did = 0 ;did < nr_devices ; did++ ) {
	    /* Allocate the cell counts and offsets. */
    if ( ( e->counts_cuda_local[did] = (int *)malloc( sizeof(int) * s->nr_cells ) ) == NULL ||
         ( e->ind_cuda_local[did] = (int *)malloc( sizeof(int) * s->nr_cells ) ) == NULL )
        return error(engine_err_malloc);
        if ( hipSetDevice( e->devices[did] ) != hipSuccess )
            return cuda_error(engine_err_cuda);
        if ( hipMalloc( &e->counts_cuda[did] , sizeof(int) * s->nr_cells ) != hipSuccess )
            return cuda_error(engine_err_cuda);
        if ( hipMalloc( &e->ind_cuda[did] , sizeof(int) * s->nr_cells ) != hipSuccess )
            return cuda_error(engine_err_cuda);
        }
        
    /* Allocate and init the taboo list on the device. */
    for ( did = 0 ;did < nr_devices ; did++ ) {
        if ( hipSetDevice( e->devices[did] ) != hipSuccess )
            return cuda_error(engine_err_cuda);
        if ( hipMalloc( &dummy[did] , sizeof(int) * s->nr_cells ) != hipSuccess )
            return cuda_error(engine_err_cuda);
        if ( hipMemset( dummy[did] , 0 , sizeof(int) * s->nr_cells ) != hipSuccess )
            return cuda_error(engine_err_cuda);
        if ( hipMemcpyToSymbol(HIP_SYMBOL( cuda_taboo ), &dummy[did] , sizeof(int *) , 0 , hipMemcpyHostToDevice ) != hipSuccess )
            return cuda_error(engine_err_cuda);
        }

    // Allocate random number generators
    if(engine_cuda_rand_norm_init(e) < 0)
        return error(engine_err);
        
    if(engine_cuda_allocate_particles(e) < 0)
        return error(engine_err);

    if(engine_cuda_load_pots(e) < 0)
        return error(engine_err);
    
    /* Init the pair queue on the device. */
    if ( engine_cuda_queues_load( e ) < 0 )
        return error(engine_err);
        
    /* He's done it! */
    return engine_err_ok;
    
    }


/**
 * @brief Removes the potentials and cell pairs on the CUDA device.
 *
 * @param e The #engine.
 *
 * @return #engine_err_ok or < 0 on error (see #engine_err).
 */
extern "C" int engine_parts_finalize(struct engine *e) {

    if(engine_cuda_rand_norm_finalize(e) < 0)
        return error(engine_err);

    if(engine_cuda_unload_pots(e) < 0)
        return error(engine_err);

    if(engine_cuda_finalize_particles(e) < 0)
        return error(engine_err);

    for(int did = 0; did < e->nr_devices; did++) {

        if(hipSetDevice(e->devices[did]) != hipSuccess)
            return cuda_error(engine_err_cuda);

        e->nrtasks_cuda[did] = 0;

        // Free the sort list, counts and indices

        if(hipFree(e->sortlists_cuda[did]) != hipSuccess)
            return cuda_error(engine_err_cuda);

        if(hipFree(e->counts_cuda[did]) != hipSuccess)
            return cuda_error(engine_err_cuda);

        if(hipFree(e->ind_cuda[did]) != hipSuccess)
            return cuda_error(engine_err_cuda);

    }

    return engine_err_ok;
}

/**
 * @brief Unload the potentials and cell pairs on the CUDA device.
 *
 * @param e The #engine.
 *
 * @return #engine_err_ok or < 0 on error (see #engine_err).
 */
 
extern "C" int engine_cuda_finalize ( struct engine *e ) {
    if(engine_parts_finalize(e) < 0)
        return error(engine_err);

    if(engine_cuda_queues_finalize(e) < 0)
        return error(engine_err);

    return engine_err_ok;
}

/**
 * @brief Refresh the engine image on the CUDA device. 
 * 
 * Can be safely called while on the CUDA device to reload all data from the engine. 
 * 
 * @param e The #engine
 * 
 * @return #engine_err_ok or < 0 on error (see #engine_err)
 */
extern "C" int engine_cuda_refresh(struct engine *e) {
    
    if(engine_cuda_finalize(e) < 0)
        return error(engine_err);

    if(engine_cuda_load(e) < 0)
        return error(engine_err);

    for(int did = 0; did < e->nr_devices; did++) {

        if(hipSetDevice(e->devices[did]) != hipSuccess)
            return cuda_error(engine_err_cuda);

        if(hipDeviceSynchronize() != hipSuccess)
            return cuda_error(engine_err_cuda);

    }

    return engine_err_ok;
}
